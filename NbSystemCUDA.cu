#include "hip/hip_runtime.h"
//
// Created by quartzar on 23/10/22.
//
#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <random>
#include <fstream>
#include <cstdio>
#include <filesystem>

#include <GL/glew.h> // glut
#include <GLFW/glfw3.h>

// lognormal distribution
#include <map>
#include <iomanip>
// #include <gsl>
#include <gsl/gsl_math.h>
#include <gsl/gsl_cdf.h>

#include "NbKernel_N2.cuh"
#include "CONSTANTS.h"
#include "NbSystemCUDA.cuh"
// #include "gnuplot-iostream.h"

/////////////////////////////////////////
// █▀█░█▀█░█▄▄░█░▀█▀░█▀▀░█▀█░░░█░█░▀▀█ //
// █▄█░█▀▄░█▄█░█░░█░░██▄░█▀▄░░░▀▄▀░░░█ //
/////////////////////////////////////////

extern __constant__ float softeningSqr;
extern __constant__ float big_G;
extern __constant__ float eta_acc;
extern __constant__ float eta_vel;

//------------PARAMETERS---------------//
NbodyRenderer::RenderMode renderMode = NbodyRenderer::POINTS;
NBodyICConfig sysConfig = NORB_SMALLN_CLUSTER;//NORB_CONFIG_SOLAR;
NbodyIntegrator integrator = LEAPFROG_VERLET;
NbodyRenderer *renderer = nullptr;
// booleans =>
bool displayEnabled = true;
bool outputBinary = true;
bool glxyCollision = false;
bool colourMode = false;
bool trailMode = false;
bool rotateCam = false;
//---------------------------------------q

/////////////////////////////////////////

//---------------------------------------
int main(int argc, char** argv)
{
    //-------------------------
    // CPU data =>
    float4 *m_hPos, *m_hVel, *m_hForce;
    float m_hDeltaTime;
    //-------------------------
    // memory transfers =>
    uint m_currentRead, m_currentWrite;
    //-------------------------
    // GPU data =>
    float4 *m_dPos[2], *m_dVel[2], *m_dForce[2];
    float *m_dDeltaTime[2];
    //-------------------------
    // OpenGL =>
    GLFWwindow *window = nullptr;
    //-------------------------
    // Timers & benchmarking =>
    auto start = std::chrono::system_clock::now();
    // std::chrono::system_clock::time_point end;
    //-------------------------
    // File output =>
    std::string outputFName = "outputCSV.csv";
    /* Binary file output */
    // std::ofstream snapshot_file;
    std::string snapshot_filename;
    std::string output_directory;

    
    //-------------------------
    // Simulation =>
    int iteration;
    int total_iterations;
    // int N_bodies;
    int snapshot_interval;
    float softening_factor;
    int snapshot_counter = 1;
    uint m_p;
    uint m_q;
    // N_orbitals = N_BODIES;
    iteration = 0;
    total_iterations = ITERATIONS;
    snapshot_interval = SNAPSHOT_INTERVAL;
    softening_factor = SOFTENING;
    // timestep = TIME_STEP;
    // float deltaTime = TIME_STEP;
    m_currentRead = 0;
    m_currentWrite = 1;
    m_p = P;
    m_q = Q;
    zoom = 1;
    ////////////////////////////////
    std::string simulation_base;
    int N_bodies;
    float softening;
    float time_start;
    float time_end;
    float current_time = 0.f;
    float delta_time;
    float snap_rate;
    float time_since_snap = 0.f;
    bool cross_time = false;
    float eta_cross;
    ////////////////////////////////
    // Read in parameters from file
    readParameters("../parameters.dat", simulation_base, N_bodies, softening, time_start,
                   time_end, snap_rate, delta_time, cross_time, eta_cross, eta_acc, eta_vel);
    std::cout << "---------------------------------" << std::endl;
    std::cout << "Simulation base: " << simulation_base << std::endl;
    std::cout << "N_bodies: " << N_bodies << std::endl;
    std::cout << "Softening: " << softening << std::endl;
    std::cout << "Time start: " << time_start << std::endl;
    std::cout << "Time end: " << time_end << std::endl;
    std::cout << "Snap rate: " << snap_rate << std::endl;
    std::cout << "Delta time: " << delta_time << std::endl;
    std::cout << "Cross time: " << cross_time << std::endl;
    std::cout << "Eta cross: " << eta_cross << std::endl;
    std::cout << "Eta acc: " << eta_acc << std::endl;
    std::cout << "Eta vel: " << eta_vel << std::endl;
    std::cout << "---------------------------------" << std::endl;
    ////////////////////////////////
    //---------------------------------------
    // INITIALISE ARRAYS & ALLOCATE DEVICE STORAGE
    //---------------------------------------
    
    // OLD / HOST
    m_hPos = new float4[N_bodies]; // x, y, z, mass
    m_hVel = new float4[N_bodies]; // vx,vy,vz, empty
    m_hForce = new float4[N_bodies]; // fx, fy, fz, empty
    // m_hDeltaTime = new float; // dt
    m_hDeltaTime = delta_time;
    // m_hDeltaTime = reinterpret_cast<float *>(TIME_STEP);
    // NEW / DEVICE
    m_dPos[0] = m_dPos[1] = nullptr;
    m_dVel[0] = m_dVel[1] = nullptr;
    m_dForce[0] = m_dForce[1] = nullptr;
    m_dDeltaTime[0] = m_dDeltaTime[1] = nullptr;
    // set memory for host arrays
    memset(m_hPos, 0, N_bodies*sizeof(float4));
    memset(m_hVel, 0, N_bodies*sizeof(float4));
    memset(m_hForce, 0, N_bodies*sizeof(float4));
    // memset(&m_hDeltaTime, 0, sizeof(float));
    getCUDAError();
    // set memory for device arrays
    allocateNOrbitalArrays(m_dPos,m_dVel, m_dForce, m_dDeltaTime, N_bodies);
    getCUDAError();
    // set device constants
    setDeviceSoftening(softening * softening);
    setDeviceBigG(1.0f * BIG_G);
    setDeviceEtaAcc(ETA_ACC);
    setDeviceEtaVel(ETA_VEL);
    getCUDAError();
    
    //---------------------------------------
    /////////////////////////////////////////
    //---------------------------------------
    
    
    // BEGIN TIMER
    runTimer(start, N_bodies, true);
    
    
    
    // INITIALISE OPENGL
    if (displayEnabled)
    {
        // glutInit(&argc, argv);
        // glutInitDisplayMode(GLUT_RGB | GLUT_DEPTH | GLUT_DOUBLE);
        window = initGL(window);
    }
    
    // Create new directory for output
    if (outputBinary) {
        // output_directory = "../out/" + getCurrentTime();
        output_directory = "../out/" + simulation_base;
        std::filesystem::create_directory(output_directory);
        if (std::filesystem::exists(output_directory)) {
            deleteFilesInDirectory(output_directory);
        } else {
            std::filesystem::create_directory(output_directory);
        }
    }
    // Set initial timestep
    // m_hDeltaTime = TIME_STEP;
    // Randomise Orbitals
    randomiseOrbitals(sysConfig, m_hPos, m_hVel, N_bodies);
    
    // Set Initial Forces [only run for solar system, HUGE performance hit]
    // if (sysConfig == NORB_CONFIG_SOLAR)
    initialiseForces(m_hPos, m_hForce, N_bodies);
    
    // Calculate and set the crossing time if needed
    if (cross_time)
    {
        float t_crossing = calculateCrossingTime(m_hVel, N_bodies);
        time_end = eta_cross * t_crossing;
        std::cout << "\n---------------------------------" << std::endl;
        std::cout << "Crossing time: " << t_crossing / 365.25 << " years" << std::endl;
        std::cout << "---------------------------------" << std::endl;
    }
    
    //---------------------------------------
    // MAIN UPDATE LOOP
    while (current_time < time_end)
    {
        if (iteration  % 10000 == 0)
            std::cout << "\nSTEP =>> " << iteration << std::flush;
    
        // Write a snapshot every snap_rate days
        if (outputBinary && time_since_snap >= snap_rate)
        {
            std::stringstream snapshot_filename_ss;
            snapshot_filename_ss << output_directory << "/snapshot_"
                                 << std::setfill('0') << std::setw(6) << std::to_string(snapshot_counter) << ".bin";
        
            snapshot_filename = snapshot_filename_ss.str();
            snapshot_counter++;
        
            writeBinaryData(snapshot_filename, current_time, m_hDeltaTime,
                            softening_factor, N_bodies, m_hPos, m_hVel, m_hForce);
            time_since_snap = 0.f;
        }
        
        simulate(m_hPos, m_dPos,
                 m_hVel, m_dVel,
                 m_hForce, m_dForce,
                 m_currentRead, m_currentWrite,
                 m_hDeltaTime, m_dDeltaTime, N_bodies, m_p, m_q);
        // std::cout << "\n m_hDeltaTime = " << m_hDeltaTime << std::flush;
        
        if (iteration % TIME_STEP_INTERVAL == 0)
        {
            m_hDeltaTime = calculateTimeStep(m_hPos, m_hVel, m_hForce, m_hDeltaTime, N_bodies);
            // std::cout << "\n m_hDeltaTime = " << m_hDeltaTime << std::flush;
        }
        
        if (displayEnabled && iteration%RENDER_INTERVAL == 0)
        {
            // CHECK FOR INPUT FIRST
            processInput(window);
    
            // CLOSE WINDOW IF ESC PRESSED
            if (glfwWindowShouldClose(window))
            {
                std::cout << "\nPROGRAM TERMINATED BY USER\nEXITING AT STEP " << iteration;
                runTimer(start,  N_bodies,false);
                finalise(m_hPos, m_dPos,
                         m_hVel, m_dVel,
                         m_hForce, m_dForce, m_dDeltaTime);
                glfwTerminate();
                exit(EXIT_SUCCESS);
            }
            
            // Render
            renderer->setPositions(reinterpret_cast<float *>(m_hPos));
            renderer->setVelocities(reinterpret_cast<float *>(m_hVel));
            renderer->display(renderMode, N_bodies, zoom, xRot, yRot, zRot, xTrans, yTrans, zTrans, trailMode, colourMode);
    
            glfwSwapBuffers(window);
            // glutSwapBuffers();
            glfwPollEvents();
    
            // Set window title to current timestep
            std::string s = std::to_string(m_hDeltaTime);//iteration);
            const char* cstr = s.c_str();
            glfwSetWindowTitle(window, cstr);
        }
        time_since_snap += m_hDeltaTime;
        current_time += m_hDeltaTime;
        iteration++;
    }
    //---------------------------------------
    
    // END TIMER
    runTimer(start,  N_bodies,false);
    
    // DELETE ARRAYS
    finalise(m_hPos, m_dPos,
             m_hVel, m_dVel,
             m_hForce, m_dForce, m_dDeltaTime);
    
    // TERMINATE SUCCESSFULLY
    glfwTerminate();
    exit(EXIT_SUCCESS);
}
//---------------------------------------

// Read in parameters from config file
//---------------------------------------
void readParameters(const std::string &filename, std::string &simulation_base, int &N_bodies, float &softening,
                    float &time_start, float &time_end, float &snap_rate, float &initial_dt,
                    bool &cross_time, float &ETA_cross, float &ETA_acc, float &ETA_vel)
{
    std::ifstream file(filename);
    
    if (file.is_open())
    {
        std::string line;
        
        while (std::getline(file, line))
        {
            std::string value;
            std::getline(file, value);
            
            if (line == "SIMULATION_BASE")
                simulation_base = value;
            else if (line == "N-BODIES")
                N_bodies = std::stoi(value);
            else if (line == "SOFTENING")
                softening = std::stof(value);
            else if (line == "TIME_START")
                time_start = std::stof(value);
            else if (line == "TIME_END")
                time_end = std::stof(value);
            else if (line == "SNAP_RATE")
                snap_rate = std::stof(value);
            else if (line == "INITIAL_DT")
                initial_dt = std::stof(value);
            else if (line == "CROSS_TIME")
            {
                if (value == "true")
                    cross_time = true;
                else if (value == "false")
                    cross_time = false;
                else
                    std::cerr << "Invalid value for CROSS_TIME: " << value << ". Use 'true' or 'false'." << std::endl;
            }
            else if (line == "ETA_CROSS")
                ETA_cross = std::stof(value);
            else if (line == "ETA_ACC")
                ETA_acc = std::stof(value);
            else if (line == "ETA_VEL")
                ETA_vel = std::stof(value);
            else
                std::cerr << "Unknown parameter: " << line << std::endl;
        }
        file.close();
    }
    else
    {
        std::cerr << "Error opening file: " << filename << std::endl;
    }
}
//---------------------------------------

// WIP: Write data to snapshot binary file
//---------------------------------------
void writeBinaryData(const std::string& filename, float current_time, float dT,
                     float softening_factor, int N, float4* pos, float4* vel, float4* force)
{
    std::ofstream file(filename, std::ios::binary | std::ios::app);
    if (file.is_open())
    {
        file.write((char*)&N, sizeof(int));
        file.write((char*)&current_time, sizeof(float));
        file.write((char*)&dT, sizeof(float));
        file.write((char*)&softening_factor, sizeof(float));
        
        for (int orbital = 0; orbital < N; orbital++)
        {
            file.write((char*)&pos[orbital].w, sizeof(float));  // Mass
            file.write((char*)&pos[orbital].x, sizeof(float));  // x position
            file.write((char*)&pos[orbital].y, sizeof(float));  // y position
            file.write((char*)&pos[orbital].z, sizeof(float));  // z position
            file.write((char*)&vel[orbital].x, sizeof(float));  // x velocity
            file.write((char*)&vel[orbital].y, sizeof(float));  // y velocity
            file.write((char*)&vel[orbital].z, sizeof(float));  // z velocity
            
            float xFrc = force[orbital].x * dT;
            float yFrc = force[orbital].y * dT;
            float zFrc = force[orbital].z * dT;
            
            file.write((char*)&xFrc, sizeof(float));    // x force
            file.write((char*)&yFrc, sizeof(float));    // y force
            file.write((char*)&zFrc, sizeof(float));    // z force
        }
        file.close();
    }
    else { std::cerr << "Error opening file: " << filename << std::endl; }
}
//---------------------------------------

// Calculate the crossing time of the system
//---------------------------------------
float calculateCrossingTime(const float4 *vel, int N)
{
    float max_v2 = 0.0f;
    for (int i = 0; i < N; i++)
    {
        float v2 = (vel[i].x * vel[i].x) + (vel[i].y * vel[i].y) + (vel[i].z * vel[i].z);
        if (v2 > max_v2) {
            max_v2 = v2;
        }
        
    }
    //     mean_v2 += (vel[i].x * vel[i].x) + (vel[i].y * vel[i].y) + (vel[i].z * vel[i].z);
    // }
    // mean_v2 /= float(N);
    
    return R_CLUSTER / std::sqrt(max_v2);
}
//---------------------------------------

// Returns the current time in the format yymmddhhmmss
//---------------------------------------
std::string getCurrentTime()
{
    auto now = std::chrono::system_clock::now();
    std::time_t now_c = std::chrono::system_clock::to_time_t(now);
    std::tm *tm = std::localtime(&now_c);
    
    std::stringstream ss;
    ss << std::put_time(tm, "%Y-%m-%d--%H-%M-%S");
    
    return ss.str();
}
//---------------------------------------



// IC generator
//---------------------------------------
void randomiseOrbitals(NBodyICConfig config, float4* pos, float4* vel, int N)
{
    using std::uniform_real_distribution;
    std::default_random_engine gen(SEED); // NOLINT(cert-msc51-cpp)
    float totalMass = 0.0;
    
    switch(config) {
        case NORB_SMALLN_CLUSTER:
        {
            float mu = 0.1;        // mean in base_10 log space, solar masses, m_0
            float sigma = 0.627;   // std. deviations in log 10 space | Chabrier, 2002
            
            // Convert the mean and standard deviation to natural logarithm space
            float ln_mu = mu * std::log(10.f);
            float ln_sigma = sigma * std::log(10.f);
    
            // Generate the lognormal distribution random masses in natural log space
            std::random_device rd;
            std::mt19937_64 rng(rd()); // mersenne-twister 19937 generator with 64-bit output
            std::lognormal_distribution<float> dist(ln_mu, ln_sigma);
    
            //  Max radius of each cluster
            // float radius = 2062.f; //10e4; // AU // 0.01 pc
            float3 cluster_centre = {0.f, 0.f, 0.f};
            float3 filament_offset = {R_CLUSTER * 2, 0.f, 0.f};
            uniform_real_distribution<float> r(-R_CLUSTER/2.f, R_CLUSTER/2.f);
            uniform_real_distribution<float> v(-1.f, 1.f); // -.1 to .1 before scaling
            
            for (int i = 0; i < N; i++)
            {
                // How many clusters? How many stars/cluster?
                // if (i % STARS_PER_CLUSTER == 0 && i > 0) { // generate new cluster
                //     cluster_centre.x += filament_offset.x;
                //     cluster_centre.y += filament_offset.y;
                //     cluster_centre.z += filament_offset.z;
                //     std::cout << "New cluster at: " << cluster_centre.x << ", " << cluster_centre.y << ", " << cluster_centre.z << std::endl;
                // }
                
                // Lognormal Initial Mass Function
                float ln_mass = dist(rng);
                float mass = std::log10(std::exp(ln_mass)); // convert back to base-10 log space
                
                // Randomised positions based on radius
                float px = r(gen);
                float py = r(gen);
                float pz = r(gen);
                
                // std::cout << "Star " << i << " at: " << px << ", " << py << ", " << pz << std::endl;
                
                // Randomised velocities
                float vx = v(gen);
                float vy = v(gen);
                float vz = v(gen);
                
                // Assign pos, vel, mass
                pos[i] = make_float4(px, py, pz, mass);
                vel[i] = make_float4(vx, vy, vz, mass);
                
                totalMass += mass;
            }
            
            // Loop through each cluster
            for (int cluster = 0; cluster < (N / STARS_PER_CLUSTER); cluster++) {
                std::cout << "Cluster " << cluster << " N: " << N << std::endl;
                int start_idx = cluster * STARS_PER_CLUSTER;
                int end_idx = start_idx + STARS_PER_CLUSTER;
    
    
                // Create a temporary vector to store positions and velocities of the cluster
                // float4 cluster_pos(STARS_PER_CLUSTER);
                float4 cluster_pos[STARS_PER_CLUSTER];
                // std::vector<float4> cluster_vel(STARS_PER_CLUSTER);
                float4 cluster_vel[STARS_PER_CLUSTER];
                
                // TODO: extract this into a function
                // Fill the temporary vector with the positions and velocities of the stars in the cluster
                for (int i = 0; i < STARS_PER_CLUSTER; i++) {
                    cluster_pos[i] = pos[start_idx + i];
                    cluster_vel[i] = vel[start_idx + i];
                    // std::cout << "StarCLUSTER " << i << " at: " << cluster_pos[i].x << ", " << cluster_pos[i].y << ", " << cluster_pos[i].z << std::endl;
                }
    
                // Apply centre of mass correction for the cluster
                float4 centreOfMassPos = calculateCentreOfMass(cluster_pos, STARS_PER_CLUSTER);
                float4 centreOfMassVel = calculateCentreOfMass(cluster_vel, STARS_PER_CLUSTER);
                for (int i = start_idx; i < end_idx; i++) {
                    // std::cout << "StarCOM " << i << " at: " << pos[i].x << ", " << pos[i].y << ", " << pos[i].z << std::endl;
                    pos[i].x -= centreOfMassPos.x;
                    pos[i].y -= centreOfMassPos.y;
                    pos[i].z -= centreOfMassPos.z;
                    vel[i].x -= centreOfMassVel.x;
                    vel[i].y -= centreOfMassVel.y;
                    vel[i].z -= centreOfMassVel.z;
                    // std::cout << "Star " << i << " at: " << pos[i].x << ", " << pos[i].y << ", " << pos[i].z << std::endl;
                    pos[i].x += filament_offset.x * float(cluster);
                    pos[i].y += filament_offset.y * float(cluster);
                    pos[i].z += filament_offset.z * float(cluster);
                    // std::cout << "StarCOM " << i << " at: " << pos[i].x << ", " << pos[i].y << ", " << pos[i].z << std::endl;
                }
                
                for (int i = 0; i < STARS_PER_CLUSTER; i++) {
                    cluster_pos[i] = pos[start_idx + i];
                    cluster_vel[i] = vel[start_idx + i];
                }
    
                // Scale the velocities to the virial theorem
                float gravitationalEnergy = -calculateGravitationalEnergy(cluster_pos, STARS_PER_CLUSTER); // W = -U
                float kineticEnergy = calculateKineticEnergy(cluster_vel, STARS_PER_CLUSTER);               // K  E = K + W | VIR = W/2 = K
    
                // float scalingFactor = sqrtf(ALPHA_VIR * gravitationalEnergy / kineticEnergy) ;
                float virialRatio = -kineticEnergy / gravitationalEnergy;
                float scalingFactor = sqrtf(.5f / virialRatio);
    
                for (int i = start_idx; i < end_idx; i++) {
                    vel[i].x *= scalingFactor;
                    vel[i].y *= scalingFactor;
                    vel[i].z *= scalingFactor;
                }
    
                for (int i = 0; i < STARS_PER_CLUSTER; i++) {
                    cluster_pos[i] = pos[start_idx + i];
                    cluster_vel[i] = vel[start_idx + i];
                }
                
                float kineticEnergyScaled = calculateKineticEnergy(cluster_vel, STARS_PER_CLUSTER);
                float verifyVirial = -kineticEnergyScaled / gravitationalEnergy;
    
                // std::cout << "Centre of mass [pos]: " << centreOfMassPos.x << ", " << centreOfMassPos.y << ", "
                //           << centreOfMassPos.z << std::endl;
                // std::cout << "Scaling factor: " << scalingFactor << std::endl;
                // std::cout << "Gravitational energy: " << gravitationalEnergy << std::endl;
                // std::cout << "Kinetic energy (unscaled): " << kineticEnergy << std::endl;
                // std::cout << "Kinetic energy (scaled): " << kineticEnergyScaled << std::endl;
                // std::cout << "Virial Ratio (initial): " << virialRatio << std::endl;
                // std::cout << "Virial Ratio (should == 0.5?): " << verifyVirial << std::endl;
            }
        }
            break;
        case NORB_CONFIG_BASIC:
        {
            uniform_real_distribution<float> randXPos(-SYS_WIDTH / 2.0, SYS_WIDTH / 2.0);
            uniform_real_distribution<float> randYPos(-SYS_HEIGHT / 2.0, SYS_HEIGHT / 2.0);
            uniform_real_distribution<float> randVel(-INIT_VEL, INIT_VEL);
            uniform_real_distribution<float> randHeight(-SYSTEM_THICKNESS, SYSTEM_THICKNESS);
            uniform_real_distribution<float> randMass(INIT_M_LOWER, INIT_M_HIGHER);
            // returns -1 to 3, so multiply by max mass/3 and clamp between min and max mass
            std::normal_distribution<float> normalDistMass(1, 0.5);
            
            
            // ASSIGNMENT LOOP
            for (int i = 0; i < N_BODIES; i++)
            {
                // getting and clamping normal distribution of mass
                const float mass = normalDistMass(gen) * ((float)INIT_M_HIGHER / 3.f);
                float massClamped;
                if (mass > 1.f * (float)INIT_M_HIGHER)
                {
                    std::cout << "\nbig boi";
                    massClamped = 100000.f;
                }
                else
                    massClamped = std::clamp(mass, (float)INIT_M_LOWER, (float)INIT_M_HIGHER);
                
                // random position assignment
                pos[i].x = randXPos(gen);
                pos[i].y = randYPos(gen);
                pos[i].z = randHeight(gen);
                pos[i].w = massClamped;
    
                // random velocity assignment
                float r = sqrtf(pos[i].x * pos[i].x + pos[i].y * pos[i].y + pos[i].z * pos[i].z);
                vel[i].x = randVel(gen) * (r / pos[i].x);//0.001f;
                vel[i].y = randVel(gen) * (r / pos[i].y);//0.001f;
                vel[i].z = 0.0f;
                vel[i].w = pos[i].w;
        
                totalMass += pos[i].w;
            }
        }
            break;
        case NORB_CONFIG_BASIC_DISK:
        {
            std::cout << "basic disk model to be implemented";
            // also to be implemented
        }
            break;
        case NORB_CONFIG_SHELL:
        {
            uniform_real_distribution<float> randF(0.0f, (float) RAND_MAX);
            uniform_real_distribution<float> randMass(INIT_M_LOWER, INIT_M_HIGHER);
    
            float scale = SYSTEM_SIZE;
            float vScale = scale * (float) VEL_SCALE / (float) KMS_TO_AUD;
            float inner = 2.5f * scale;
            float outer = 4.0f * scale;
    
            pos[0].x = 0.0;
            pos[0].y = 0.0;
            pos[0].z = 0.0;
            pos[0].w = CENTRE_STAR_M;
    
            vel[0].x = 0.0;
            vel[0].y = 0.0;
            vel[0].z = 0.0;
            vel[0].w = CENTRE_STAR_M;
    
    
            int i = 1;
            while (i < N_BODIES) {
                float x, y, z;
                x = randF(gen) / (float) RAND_MAX * 2.0f - 1.0f;
                y = randF(gen) / (float) RAND_MAX * 2.0f - 1.0f;
                z = randF(gen) / (float) RAND_MAX * 2.0f - 1.0f;
        
                float3 point = {x, y, z};
                float len = normalise(point);
                if (len > 1)
                    continue;
    
                pos[i].x = point.x * (inner + (outer - inner) * randF(gen) / (float) RAND_MAX);
                pos[i].y = point.x * (inner + (outer - inner) * randF(gen) / (float) RAND_MAX);
                pos[i].z = point.x * (inner + (outer - inner) * randF(gen) / (float) RAND_MAX);
                pos[i].w = randMass(gen);
                
        
                x = 0.0f;
                y = 0.0f;
                z = 1.0f;
        
                float3 axis = {x, y, z};
                normalise(axis);
        
                if (1 - dot(point, axis) < 1e-6) {
                    axis.x = point.y;
                    axis.y = point.x;
                    normalise(axis);
                }
                float3 vv = {pos[i].x, pos[i].y, pos[i].z};
                vv = cross(vv, axis);
                vel[i].x = vv.x * vScale;
                vel[i].y = vv.y * vScale;
                vel[i].z = vv.z * vScale;
                vel[i].w = pos[i].w;
        
                i++;
            }
        }
            break;
        case NORB_CONFIG_EXPAND:
        {
            uniform_real_distribution<float> randF(0.0f, (float) RAND_MAX);
            uniform_real_distribution<float> randMass(INIT_M_LOWER, INIT_M_HIGHER);
    
            float scale = SYSTEM_SIZE * std::max(1.0f, (float)N / (1024.f));
            float vScale = scale * (float) VEL_SCALE / (float) KMS_TO_AUD;
            
            for (int i = 0; i < N;)
            {
                float3 point;
                point.x = randF(gen) / (float) RAND_MAX * 2.0f - 1.0f;
                point.y = randF(gen) / (float) RAND_MAX * 2.0f - 1.0f;
                point.z = randF(gen) / (float) RAND_MAX * 2.0f - 1.0f;
                
                float lengthSq = dot(point, point);
                if (lengthSq > 1)
                    continue;
    
                pos[i].x = point.x * scale;
                pos[i].y = point.y * scale;
                pos[i].z = point.z * scale;
                pos[i].w = randMass(gen);
                vel[i].x = point.x * vScale; //* float(PI)/180 * lengthSq;
                vel[i].y = point.y * vScale;
                vel[i].z = point.z * vScale;
                vel[i].w = pos[i].w;
                
                i++;
            }
            
        }
            break;
        case NORB_CONFIG_ADV_DISK:
        {
            // uniform_real_distribution<float> randF(0.0f, (float) RAND_MAX);
            // uniform_real_distribution<float> randMass(0.0, 5);
            uniform_real_distribution<float> randMassInner(ADVD_M_INNER_MIN, ADVD_M_INNER_MAX);
            // uniform_real_distribution<float> randMassOuter(INIT_M_LOWER, INIT_M_HIGHER);
            
            pos[0].x = 0.0;
            pos[0].y = 0.0;
            pos[0].z = 0.0;
            pos[0].w = ADVD_CENTRE_M;
    
            vel[0].x = 0.0;
            vel[0].y = 0.0;
            vel[0].z = 0.0;
            vel[0].w = ADVD_CENTRE_M;
    
            float c      = ADVD_C_INNER; // flatness
            float mass   = randMassInner(gen);
            // float mass = randMassInner(gen);
            float radius = ADVD_R_INNER;
            
            int start;
            if (glxyCollision)
            {
                pos[1].x = 1000.0;
                pos[1].y = 500.0;
                pos[1].z = -10000.0;
                pos[1].w = ADVD_G2_MASS;
    
                vel[1].x = -0.1;
                vel[1].y = 0.0;
                vel[1].z = 1.0;
                vel[1].w = ADVD_G2_MASS;
                start = 2;
            }
            else
                start = 1;
            for (int i = start;i < N; i++)
            {
                if (i == N - ADVD_OUTER_N) {
                    c = ADVD_C_OUTER;
                    mass = ADVD_M_OUTER;
                    // mass = randMassOuter(gen) * 100.0f;
                    radius = ADVD_R_OUTER;
                }
                
                float3 position;
                while (true)
                {
                    position.x = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
                    position.y = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
                    
                    if (position.y >= -1.0f * sqrtf(1.0f - powf(position.x, 2.0f))
                        && position.y <= sqrtf(1.0f - powf(position.x, 2.0f)))
                        break;
                }
                
                float zPosMax = sqrtf(c * (1.0f - powf(position.x, 2.0f)
                        -powf(position.y, 2.0f)));
                float zPosMin = -1.0f * zPosMax;
                float zPosRand = rand() / (float) RAND_MAX;
                position.z = (zPosMax - zPosMin) * zPosRand + zPosMin;
    
                position.x *= radius;
                position.y *= radius;
                position.z *= radius;
                
                float m = position.y / position.x;
                m = -1.0f / m;
                float b = position.y - position.x * m;
                
                float3 velocity;
                // float vel_m = sqrtf(((float)BIG_G * (1e6f + mass * 1.2e-6f)) /
                //                     sqrtf(position.x*position.x + position.y*position.y + position.z*position.z));
                float vel_m = sqrtf(((float)BIG_G * (ADVD_CENTRE_M + mass * 1.2e2f)) /
                                    sqrtf(position.x * position.x + position.y * position.y + position.z * position.z));
                
                if (position.y > 0)
                {
                    velocity = {-1.0f * (radius / 2.0f), (position.x - radius / 2.0f) * m + b - position.y, 0};
                    vel_m /= sqrtf(velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z);
                    velocity.x *= vel_m;
                    velocity.y *= vel_m;
                    velocity.z *= vel_m;
                    
                }
                else
                {
                    velocity = {(radius / 2.0f), (position.x + radius / 2.0f) * m + b - position.y, 0};
                    vel_m /= sqrtf(velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z);
                    velocity.x *= vel_m;
                    velocity.y *= vel_m;
                    velocity.z *= vel_m;
                }
                
                float pScale = 1.0f;
                pos[i].x = position.x * pScale;
                pos[i].y = position.y * pScale;
                pos[i].z = position.z * pScale;
                pos[i].w = mass;
    
                vel[i].x = velocity.x;
                vel[i].y = velocity.y;
                vel[i].z = velocity.z;
                vel[i].w = mass;
                
                // std::cout << "\n " << velocity[i].x << " " << velocity[i].y << " " << velocity[i].z;
    
            }
        }
            break;
        case NORB_CONFIG_ADV_DISK_COLLSION:
        {
            // hi
            uniform_real_distribution<float> randMassInner(ADVD_M_INNER_MIN, ADVD_M_INNER_MAX);
    
            pos[0].x = 0.0;
            pos[0].y = 0.0;
            pos[0].z = 0.0;
            pos[0].w = ADVD_CENTRE_M;
    
            vel[0].x = 0.0;
            vel[0].y = 0.0;
            vel[0].z = 0.0;
            vel[0].w = ADVD_CENTRE_M;
    
            pos[N / 2].x = ADVD_G2_X;
            pos[N / 2].y = ADVD_G2_Y;
            pos[N / 2].z = ADVD_G2_Z;
            pos[N / 2].w = ADVD_CENTRE_M;
    
            vel[N / 2].x = -1.f * ADVD_G2_VX;
            vel[N / 2].y = -1.f * ADVD_G2_VY;
            vel[N / 2].z = -1.f * ADVD_G2_VZ;
            vel[N / 2].w = ADVD_CENTRE_M;
    
            float c      = ADVD_C_INNER; // flatness
            float mass;//   = randMassInner(gen);
            // float mass = randMassInner(gen);
            float radius = ADVD_R_INNER;
    
            int start = 1;
            for (int i = start;i < N/2; i++)
            {
                mass = randMassInner(gen);
                if (i == N/2 - ADVD_OUTER_N) {
                    c = ADVD_C_OUTER;
                    mass = ADVD_M_OUTER;
                    // mass = randMassOuter(gen) * 100.0f;
                    radius = ADVD_R_OUTER;
                }
        
                float3 position;
                while (true)
                {
                    position.x = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
                    position.y = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
            
                    if (position.y >= -1.0f * sqrtf(1.0f - powf(position.x, 2.0f))
                        && position.y <= sqrtf(1.0f - powf(position.x, 2.0f)))
                        break;
                }
        
                float zPosMax = sqrtf(c * (1.0f - powf(position.x, 2.0f)
                                           -powf(position.y, 2.0f)));
                float zPosMin = -1.0f * zPosMax;
                float zPosRand = rand() / (float) RAND_MAX;
                position.z = (zPosMax - zPosMin) * zPosRand + zPosMin;
    
                position.x *= radius;
                position.y *= radius;
                position.z *= radius;
        
                float m = position.y / position.x;
                m = -1.0f / m;
                float b = position.y - position.x * m;
        
                float3 velocity;
                // float vel_m = sqrtf(((float)BIG_G * (1e6f + mass * 1.2e-6f)) /
                //                     sqrtf(position.x*position.x + position.y*position.y + position.z*position.z));
                float vel_m = sqrtf(((float)BIG_G * (ADVD_CENTRE_M + mass * 1.2e2f)) /
                                    sqrtf(position.x * position.x + position.y * position.y + position.z * position.z));
        
                if (position.y > 0)
                {
                    velocity = {-1.0f * (radius / 2.0f), (position.x - radius / 2.0f) * m + b - position.y, 0};
                    vel_m /= sqrtf(velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z);
                    velocity.x *= vel_m;
                    velocity.y *= vel_m;
                    velocity.z *= vel_m;
            
                }
                else
                {
                    velocity = {(radius / 2.0f), (position.x + radius / 2.0f) * m + b - position.y, 0};
                    vel_m /= sqrtf(velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z);
                    velocity.x *= vel_m;
                    velocity.y *= vel_m;
                    velocity.z *= vel_m;
                }
        
                float pScale = 1.0f;
                pos[i].x = position.x * pScale;
                pos[i].y = position.y * pScale;
                pos[i].z = position.z * pScale;
                pos[i].w = mass;
    
                vel[i].x = velocity.x;
                vel[i].y = velocity.y;
                vel[i].z = velocity.z;
                vel[i].w = mass;
                }
    
            c      = ADVD_C_INNER; // flatness
            mass   = randMassInner(gen);
            // float mass = randMassInner(gen);
            radius = ADVD_R_INNER;
            start = N/2 + 1;
            for (int i = start;i < N; i++)
            {
                mass = randMassInner(gen);
                if (i == N - ADVD_OUTER_N) {
                    c = ADVD_C_OUTER;
                    mass = ADVD_M_OUTER;
                    // mass = randMassOuter(gen) * 100.0f;
                    radius = ADVD_R_OUTER;
                }
        
                float3 position;
                while (true)
                {
                    position.x = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
                    position.y = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
            
                    if (position.y >= -1.0f * sqrtf(1.0f - powf(position.x, 2.0f))
                        && position.y <= sqrtf(1.0f - powf(position.x, 2.0f)))
                        break;
                }
        
                float zPosMax = sqrtf(c * (1.0f - powf(position.x, 2.0f)
                                           -powf(position.y, 2.0f)));
                float zPosMin = -1.0f * zPosMax;
                float zPosRand = rand() / (float) RAND_MAX;
                position.z = (zPosMax - zPosMin) * zPosRand + zPosMin;
    
                position.x *= radius;
                position.y *= radius;
                position.z *= radius;
        
                float m = position.y / position.x;
                m = -1.0f / m;
                float b = position.y - position.x * m;
        
                float3 velocity;
                // float vel_m = sqrtf(((float)BIG_G * (1e6f + mass * 1.2e-6f)) /
                //                     sqrtf(position.x*position.x + position.y*position.y + position.z*position.z));
                float vel_m = sqrtf(((float)BIG_G * (ADVD_CENTRE_M + mass * 1.2e2f)) /
                                    sqrtf(position.x * position.x + position.y * position.y + position.z * position.z));
        
                if (position.y > 0)
                {
                    velocity = {-1.0f * (radius / 2.0f), (position.x - radius / 2.0f) * m + b - position.y, 0};
                    vel_m /= sqrtf(velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z);
                    velocity.x *= vel_m;
                    velocity.y *= vel_m;
                    velocity.z *= vel_m;
            
                }
                else
                {
                    velocity = {(radius / 2.0f), (position.x + radius / 2.0f) * m + b - position.y, 0};
                    vel_m /= sqrtf(velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z);
                    velocity.x *= vel_m;
                    velocity.y *= vel_m;
                    velocity.z *= vel_m;
                }
    
    
                pos[i].x = position.z + (float)ADVD_G2_X;
                pos[i].y = position.y + (float)ADVD_G2_Y;
                pos[i].z = position.x + (float)ADVD_G2_Z;
                pos[i].w = mass;
    
                vel[i].x = velocity.z - (float)ADVD_G2_VX;
                vel[i].y = velocity.y - (float)ADVD_G2_VY;
                vel[i].z = velocity.x - (float)ADVD_G2_VZ;
                vel[i].w = mass;
            }
            
        }
            break;
        case NORB_CONFIG_SOLAR:
        {
            int i = 0;
            // The Sun
            pos[i].x = pos[i].y = pos[i].z = 0.f;
            pos[i].w = 1.f;
    
            vel[i].x = vel[i].y = vel[i].z = 0.f;
            vel[i].w = 1.f;
    
            // Earth
            pos[++i].x = 1.f;
            pos[i].y = 0.f;
            pos[i].z = 0.f;
            pos[i].w = 3.00273e-6f;// 2.9861e-6f;
    
            vel[i].x = 0.f;
            vel[i].y = 29.795f / KMS_TO_AUD;//29.78f / (float)KMS_TO_AUD;
            vel[i].z = 0.f;
            vel[i].w = 3.00273e-6f;
    
            // // Mercury
            // pos[++i] = {.387f, 0.f, 0.f, 1.651e-7f};
            // vel[i]   = {0.f, 47.36f/KMS_TO_AUD, 0.f, 1.651e-7f};
            //
            // // Venus
            // pos[++i].x = 0.723f;
            // pos[i].y = 0.f;
            // pos[i].z = 0.f;
            // pos[i].w = 2.447e-6f;
            //
            // vel[i].x = 0.f;
            // vel[i].y = 35.02f / KMS_TO_AUD;
            // vel[i].z = 0.f;
            // vel[i].w = 2.447e-6f;
            //
            // // Mars
            // pos[++i] = {1.524f, 0.f, 0.f, 3.213e-7f};
            // vel[i]   = {0.f, 24.07f/KMS_TO_AUD, 0.f, 3.213e-7f};
            
        }
            break;
    }
    std::cout << "\nTOTAL MASS ->> " << totalMass;
}
//---------------------------------------

// Calculate Centre of Mass
//---------------------------------------
float4 calculateCentreOfMass(float4* body, int N)
{
    float4 centreOfMass = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    for (int i = 0; i < N; i++)
    {
        centreOfMass.x += body[i].x * body[i].w;
        centreOfMass.y += body[i].y * body[i].w;
        centreOfMass.z += body[i].z * body[i].w;
        centreOfMass.w += body[i].w;
    }
    centreOfMass.x /= centreOfMass.w;
    centreOfMass.y /= centreOfMass.w;
    centreOfMass.z /= centreOfMass.w;
    return centreOfMass;
}
//---------------------------------------


// Calculate Gravitational Energy
//---------------------------------------
float calculateGravitationalEnergy(float4* pos, int N)
{
    float gravitationalEnergy = 0.0f;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            if (i==j)
                continue;
            
            float3 r;
            r.x = pos[j].x - pos[i].x;
            r.y = pos[j].y - pos[i].y;
            r.z = pos[j].z - pos[i].z;
            
            float distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
            
            gravitationalEnergy += float(BIG_G) * pos[i].w * pos[j].w / sqrtf(distSqr);
            // std::cout << "Distance Sqr ->> " << distSqr << std::endl;
            // std::cout << "Distance Sqrt ->> " << sqrtf(distSqr) << std::endl;
            // std::cout << "Gravitational Energy ->> " << gravitationalEnergy << std::endl;
        }
    }
    return gravitationalEnergy * .5f;   // .5f because each pairwise interaction is counted twice
}
//---------------------------------------


// Calculate Kinetic Energy
//---------------------------------------
float calculateKineticEnergy(float4* vel, int N)
{
    float kineticEnergy = 0.0f;
    for (int i = 0; i < N; i++)
    {
        kineticEnergy += 0.5f * vel[i].w * (vel[i].x * vel[i].x + vel[i].y * vel[i].y + vel[i].z * vel[i].z);
    }
    return kineticEnergy;
}
//---------------------------------------



// Initialise Forces [typically unnecessary unless for solar-system]
//---------------------------------------
void initialiseForces(float4* pos, float4* force, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            if (i == j)
                continue;
            
            float3 r;
    
            // r_ij -> AU [distance]
            r.x = pos[j].x - pos[i].x;
            r.y = pos[j].y - pos[i].y;
            r.z = pos[j].z - pos[i].z;
    
            // distance squared == dot(r_ij, r_ij) + softening^2
            float distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
            distSqr += SOFTENING * SOFTENING;
    
            // inverse distance cubed == 1 / distSqr^(3/2) [fastest method]
            float distSixth = distSqr * distSqr * distSqr;
            float invDistCube = 1.0f / sqrtf(distSixth);
    
            // force = mass_j * inverse distance cube
            float f = pos[j].w * invDistCube;
    
            // acceleration = acceleration_i + force * r_ij
            force[i].x += r.x * f * (float)BIG_G;
            force[i].y += r.y * f * (float)BIG_G;
            force[i].z += r.z * f * (float)BIG_G;
        }
    }
}
//---------------------------------------


// Calculate variable time-step
//---------------------------------------
float calculateTimeStep(float4* pos, float4* vel, float4* force, float curDT, int N)
{
    auto* acc_dot = new float3[N];
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            if (i == j) continue;
            
            float3 r; float3 v;
            // r_ij -> AU [distance]
            r.x = pos[j].x - pos[i].x;
            r.y = pos[j].y - pos[i].y;
            r.z = pos[j].z - pos[i].z;
            
            // v_ij -> AU/day [velocity]
            v.x = vel[j].x - vel[i].x;
            v.y = vel[j].y - vel[i].y;
            v.z = vel[j].z - vel[i].z;
            
            // distance squared == dot(r_ij, r_ij) + softening^2
            float distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
            float dist = sqrtf(distSqr);
            distSqr += SOFTENING * SOFTENING;
            
            // inverse distance cubed == 1 / distSqr^(3/2) [fastest method]
            float distSixth = distSqr * distSqr * distSqr;
            float invDistCube = 1.0f / sqrtf(distSixth);
            float distFifth = distSqr * distSqr * dist;
            float invDistFifth = 1.0f / sqrtf(distFifth);
            
            // force = mass_j * inverse distance cube
            float f1 = pos[j].w * invDistCube;
            float f2 = pos[j].w * invDistFifth;
            
            float vdotr = dot(v, r);
            
            // acceleration
            // acc_dot[i].x += ( (v.x * f1) + ((3 * v.x * r.x) * r.x * f2) ) * (float)BIG_G;
            acc_dot[i].x += ( (v.x * f1) + ((3 * vdotr) * r.x * f2) ) * (float)BIG_G;
            acc_dot[i].y += ( (v.y * f1) + ((3 * vdotr) * r.y * f2) ) * (float)BIG_G;
            acc_dot[i].z += ( (v.z * f1) + ((3 * vdotr) * r.z * f2) ) * (float)BIG_G;
        }
    }
    
    float max_aa_dot = 0.0f;
    float max_a = 0;
    float max_v = 0.0f;
    for (int i = 0; i < N; i++)
    {
        float a_dot = acc_dot[i].x * acc_dot[i].x + acc_dot[i].y * acc_dot[i].y + acc_dot[i].z * acc_dot[i].z;
        float a = force[i].x * force[i].x + force[i].y * force[i].y + force[i].z * force[i].z;
        float v = vel[i].x * vel[i].x + vel[i].y * vel[i].y + vel[i].z * vel[i].z;
        a_dot = sqrtf(a_dot);
        a = sqrtf(a);
        v = sqrtf(v);
        
        float aa_dot = a / a_dot;
        max_aa_dot = (aa_dot > max_aa_dot) ? aa_dot : max_aa_dot;
        max_a = (a > max_a) ? a : max_a;
        max_v = (v > max_v) ? v : max_v;
    }
    float dt = ETA_ACC * max_aa_dot + ETA_VEL * (max_v / max_a);
    return fminf(dt, MAX_DELTA_TIME);
}

//---------------------------------------

// float calculateTimeStep(float4* pos, float4* vel, float4* force, float curDT, int N)
// {
//     float maxAcc = 0.0f;
//     int maxAccIndex = 0;
//     float maxVel = 0.0f;
//     for (int i = 0; i < N; i++)
//     {
//         // Acceleration
//         float accel = force[i].x * force[i].x + force[i].y * force[i].y + force[i].z * force[i].z;
//         accel /= pos[i].w;
//         float accelMag = sqrtf(accel);
//
//         maxAccIndex = (accelMag > maxAcc) ? i : maxAccIndex;
//         maxAcc = (accelMag > maxAcc) ? accelMag : maxAcc;
//
//         // Velocity
//         float velocity = vel[i].x * vel[i].x + vel[i].y * vel[i].y + vel[i].z * vel[i].z;
//         float velMag = sqrtf(velocity);
//
//         maxVel = (velMag > maxVel) ? velMag : maxVel;
//     }
//     float m = pos[maxAccIndex].w;
//     float3 a = make_float3(force[maxAccIndex].x / m,
//                            force[maxAccIndex].y / m,
//                            force[maxAccIndex].z / m);
//     float aDot = dot(a, a);
//     float dt = ETA_ACC * maxAcc / aDot + ETA_VEL * maxVel / curDT * maxAcc;
//     std::cout << "\nDelta Time ->> " << dt << std::endl;
//     return fminf(dt, MAX_DELTA_TIME);
// }

//---------------------------------------
// MAIN UPDATE LOOP
//---------------------------------------
void simulate(float4* m_hPos, float4* m_dPos[2],
              float4* m_hVel, float4* m_dVel[2],
              float4* m_hForce, float4* m_dForce[2],
              uint m_currentRead, uint m_currentWrite,
              float& m_hDeltaTime, float* m_dDeltaTime[2], int N, uint m_p, uint m_q)
{
    // set pos,vel -> update -> get pos,vel ~@ repeat
    
    // Send data to device first
    copyDataToDevice(m_dPos[m_currentRead], m_hPos, N);
    copyDataToDevice(m_dVel[m_currentRead], m_hVel, N);
    copyDataToDevice(m_dForce[m_currentRead], m_hForce, N);
    hipMemcpy(m_dDeltaTime[m_currentRead], &m_hDeltaTime, sizeof(float), hipMemcpyHostToDevice);
    getCUDAError();


    // Do the thing
    deployToGPU(m_dPos[m_currentRead], m_dPos[m_currentWrite],
                m_dVel[m_currentRead], m_dVel[m_currentWrite],
                m_dForce[m_currentRead], m_dForce[m_currentWrite],
                m_dDeltaTime[m_currentRead], m_dDeltaTime[m_currentWrite], N, m_p, m_q);
    // Swap read and write
    std::swap(m_currentRead, m_currentWrite);

    hipDeviceSynchronize();
    
    // Retrieve data from device
    copyDataToHost(m_hPos, m_dPos[m_currentRead], N);
    copyDataToHost(m_hVel, m_dVel[m_currentRead], N);
    copyDataToHost(m_hForce, m_dForce[m_currentRead], N);
    hipMemcpy(&m_hDeltaTime, m_dDeltaTime[m_currentRead], sizeof(float), hipMemcpyDeviceToHost);
    getCUDAError();
    // std::cout << "Time step: " << m_hDeltaTime << std::endl;
    // Retrieve any CUDA errors and output
    getCUDAError();
}
//---------------------------------------


// CUDA error check
//---------------------------------------
void getCUDAError()
{
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) std::cout << "\nCUDA error:%s\n" << hipGetErrorString(hipError_t);
}
//---------------------------------------


// Finalise & delete arrays TODO: reimplement this
//---------------------------------------
void finalise(float4* m_hPos, float4* m_dPos[2],
              float4* m_hVel, float4* m_dVel[2],
              float4* m_hForce, float4* m_dForce[2],
              float* m_dDeltaTime[2])
{
    delete [] m_hPos;
    delete [] m_hVel;
    delete [] m_hForce;
    // delete m_hDeltaTime;
    
    deleteNOrbitalArrays(m_dPos, m_dVel, m_dForce, m_dDeltaTime);
}
//---------------------------------------


// A nice little normalisation function
//---------------------------------------
float normalise(float3& vector)
{
    float dist = sqrtf(vector.x*vector.x + vector.y*vector.y + vector.z*vector.z);
    if (dist > 1e-6)
    {
        vector.x /= dist;
        vector.y /= dist;
        vector.z /= dist;
    }
    return dist;
}
//---------------------------------------

// Deletes snapshot folder files if already exists
//---------------------------------------
void deleteFilesInDirectory(const std::string& directory_path)
{
    try
    {
        for (const auto &entry : std::filesystem::directory_iterator(directory_path))
        {
            std::filesystem::remove(entry.path());
        }
    }
    catch (std::filesystem::filesystem_error &e)
    {
        std::cerr << "Error deleting files in directory: " << e.what() << std::endl;
    }
}
//---------------------------------------


//////////////////////////////////////////////////////
// █▀▀ █▀█ █░█   █▀▀ █░█ █▄░█ █▀▀ ▀█▀ █ █▀█ █▄░█ █▀ //
// █▄█ █▀▀ █▄█   █▀░ █▄█ █░▀█ █▄▄ ░█░ █ █▄█ █░▀█ ▄█ //
//////////////////////////////////////////////////////

extern "C"
{

// Send softening_sqr value to device
//---------------------------------------
void setDeviceSoftening(float softening)
{
    float softeningSq = softening * softening;
    
    hipMemcpyToSymbol(HIP_SYMBOL(softeningSqr), &softeningSq, sizeof(float),0);
}
//---------------------------------------


// Send gravitational constant to device
//---------------------------------------
void setDeviceBigG(float G)
{
    hipMemcpyToSymbol(HIP_SYMBOL(big_G), &G, sizeof(float),0);
}
//---------------------------------------


// Send eta_acc value to device
//---------------------------------------
void setDeviceEtaAcc(float eta)
{
    hipMemcpyToSymbol(HIP_SYMBOL(eta_acc), &eta, sizeof(float),0);
}

// Send eta_vel value to device
//---------------------------------------
void setDeviceEtaVel(float eta)
{
    hipMemcpyToSymbol(HIP_SYMBOL(eta_vel), &eta, sizeof(float),0);
}


// Allocate device memory for variables
//---------------------------------------
void allocateNOrbitalArrays(float4* pos[2], float4* vel[2], float4* force[2], float* dT[2],  int N)
{
    // memory size for device allocation
    uint memSize = sizeof(float4) * N;
    
    hipMalloc((void**)&pos[0], memSize);
    hipMalloc((void**)&pos[1], memSize);
    hipMalloc((void**)&vel[0], memSize);
    hipMalloc((void**)&vel[1], memSize);
    hipMalloc((void**)&force[0], memSize);
    hipMalloc((void**)&force[1], memSize);
    hipMalloc((void**)&dT[0], sizeof(float));
    hipMalloc((void**)&dT[1], sizeof(float));
}
//---------------------------------------


// De-allocate device memory variables
//---------------------------------------
void deleteNOrbitalArrays(float4* pos[2], float4* vel[2], float4* force[2], float* dT[2])
{
    hipFree((void**)pos[0]);
    hipFree((void**)pos[1]);
    hipFree((void**)vel[0]);
    hipFree((void**)vel[1]);
    hipFree((void**)force[0]);
    hipFree((void**)force[1]);
    hipFree((void**)dT[0]);
    hipFree((void**)dT[1]);
}
//---------------------------------------


// Copy data from host[CPU] ->> device[GPU]
//---------------------------------------
void copyDataToDevice(float4* device, const float4* host, int N)
{
    uint memSize = sizeof(float4) * N;
    hipMemcpy(device, host, memSize, hipMemcpyHostToDevice);
    getCUDAError();
}
//---------------------------------------


// Copy data from device[GPU] ->> host[CPU]
//---------------------------------------
void copyDataToHost(float4* host, const float4* device, int N)
{
    uint memSize = sizeof(float4) * N;
    hipMemcpy(host, device, memSize, hipMemcpyDeviceToHost);
    getCUDAError();
}
//---------------------------------------


// Initiates GPU kernel computations every iteration
//---------------------------------------
void deployToGPU(float4* oldPos, float4* newPos,
                 float4* oldVel, float4* newVel,
                 float4* oldForce, float4* newForce,
                 float* oldDT, float* newDT, int N, uint p, uint q)
{
    uint shMemSize = p * q * sizeof(float4);
    
    // thread and grid time :D
    dim3 threads(p, q, 1);
    dim3 grid(N / p, 1, 1);
    
    // DEPLOY TODO: removed feature
    /*If multithreading is enabled (i.e. q>1 | multiple threads per
     * body) then the more complicated code is executed (using bool template
     * over in the kernel), and if it is not, then the simpler code is executed*/

    switch(integrator)
    {
        case LEAPFROG_VERLET:
        default:
        {
            integrateNOrbitals<<<grid, threads, shMemSize
            >>>(oldPos, newPos, oldVel, newVel, oldForce, newForce, oldDT, newDT, N);
        }
        break;
        case KICK_DRIFT_VERLET:
        {
            initHalfKickForces<<<grid, threads, shMemSize
            >>>(oldPos, newPos, oldVel, newVel, oldForce, newForce, oldDT, N);
            hipDeviceSynchronize();
            fullKickForces<<<grid, threads, shMemSize
            >>>(oldPos, newPos, oldVel, newVel, oldForce, newForce, oldDT, N);
        }
        break;
    }
}
//---------------------------------------
}

// MISC FUNCTIONS

// Timer, very simple
//---------------------------------------
void runTimer(std::chrono::system_clock::time_point start,
              int N_orbitals, bool init)
{
    if (init)
    {
        start = std::chrono::system_clock::now();
        std::time_t start_time = std::chrono::system_clock::to_time_t(start);
        std::cout << "Starting Simulation at ->> " << std::ctime(&start_time)
                  << "For N == " << N_orbitals << " || Iterations == " << ITERATIONS << std::endl;
    }
    else // end
    {
        auto end = std::chrono::system_clock::now();
        std::chrono::duration<double> elapsed_seconds = end-start;
        std::time_t end_time = std::chrono::system_clock::to_time_t(end);
        std::cout << "\nFinished Computation at ->> " << std::ctime(&end_time)
                  << "Elapsed Time : " << elapsed_seconds.count() << "s"
                  << " for N = " << N_orbitals << std::endl;
    }
}
//---------------------------------------


// Initialise OpenGL for particle rendering
//---------------------------------------
GLFWwindow* initGL(GLFWwindow *window)
{
    if(!glewInit())
    {
        std::cout << "\nCritical OpenGL error ::\nFailed to initialise GLEW\nTERMINATING";
        glfwTerminate();
        exit(EXIT_FAILURE);
    }
    if (!glfwInit())
    {   // SAFETY CHECK
        std::cout << "\nCritical OpenGL error ::\nFailed to initialise GLFW\nTERMINATING";
        glfwTerminate();
        exit(EXIT_FAILURE);
    }
    
    // CREATE WINDOW IN WINDOWED MODE
    glfwWindowHint(GLFW_RESIZABLE, GL_TRUE);
    window = glfwCreateWindow(WIDTH, HEIGHT, "orbiterV6", nullptr, nullptr);
    
    if (!window)
    {   // SAFETY CHECK
        std::cout << "\n Critical OpenGL error ::\nFailed to open GLFW window\nTERMINATING";
        glfwTerminate();
        exit (EXIT_FAILURE);
    }
    // CALLBACKS
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback); // -> viewport
    glfwSetKeyCallback(window, key_callback); // -> key input
    glfwSetScrollCallback(window, scroll_callback); // -> scroll input
    
    // set window context | synchronise to refresh rate with swapinterval
    glfwMakeContextCurrent(window);
    
    // SET THE VIEWPORT
    glViewport(0, 0, WIDTH, HEIGHT);
    // SET THE PROJECTION TRANSFORM
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(FOV, (GLfloat)WIDTH/(GLfloat)HEIGHT, 0, V_FAR); // TODO: rename to Z_FAR
    
    // PREPARE WINDOW
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GL_LEQUAL); // experimental
    glClearColor(0.0, 0.0, 0.0, 1.0);
    
    // PREPARE RENDERER
    renderer = new NbodyRenderer;
    
    // TODO: add GL error check here
    return window;
}
//---------------------------------------


// A nice little vector cross product function
//---------------------------------------
float3 cross(float3 v0, float3 v1)
{
    float3 rt;
    rt.x = v0.y*v1.z-v0.z*v1.y;
    rt.y = v0.z*v1.x-v0.x*v1.z;
    rt.z = v0.x*v1.y-v0.y*v1.x;
    return rt;
}
//---------------------------------------


// A nice little vector dot product function
//---------------------------------------
float dot(float3 v0, float3 v1)
{
    return v0.x*v1.x+v0.y*v1.y+v0.z*v1.z;
}
//---------------------------------------


// Processes user input for sim control
//---------------------------------------
void processInput(GLFWwindow *window)
{
    // if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
    
    if (glfwGetKey(window, GLFW_KEY_LEFT_SHIFT) == GLFW_PRESS)
        shiftSpeed = 1 * SHIFT_FACTOR;
    if (glfwGetKey(window, GLFW_KEY_LEFT_SHIFT) == GLFW_RELEASE)
        shiftSpeed = 1;
    // if (glfwGetKey(window, GLFW_KEY_LEFT_CONTROL) == GLFW_PRESS)
    //     shiftSpeed = 1 * CTRL_FACTOR;
    // if (glfwGetKey(window, GLFW_KEY_LEFT_CONTROL) == GLFW_RELEASE)
    //     shiftSpeed = 1;
    if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS)
        zTrans += shiftSpeed * MOVE_SPEED * 1.0f;
    if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS)
        zTrans -= shiftSpeed * MOVE_SPEED * 1.0f;
    if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS)
        xTrans += shiftSpeed * MOVE_SPEED;
    if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS)
        xTrans -= shiftSpeed * MOVE_SPEED;
    if (glfwGetKey(window, GLFW_KEY_SPACE) == GLFW_PRESS)
        yTrans -= shiftSpeed * MOVE_SPEED;
    if (glfwGetKey(window, GLFW_KEY_LEFT_CONTROL) == GLFW_PRESS)
        yTrans += shiftSpeed * MOVE_SPEED;
    if (glfwGetKey(window, GLFW_KEY_LEFT) == GLFW_PRESS)
        xRot += shiftSpeed * 1;
    if (glfwGetKey(window, GLFW_KEY_RIGHT) == GLFW_PRESS)
        xRot -= shiftSpeed * 1;
    if (glfwGetKey(window, GLFW_KEY_UP) == GLFW_PRESS)
        yRot += shiftSpeed * 1;
    if (glfwGetKey(window, GLFW_KEY_DOWN) == GLFW_PRESS)
        yRot -= shiftSpeed * 1;
    if (glfwGetKey(window, GLFW_KEY_Z) == GLFW_PRESS)
        zRot += shiftSpeed * 1;
    if (glfwGetKey(window, GLFW_KEY_X) == GLFW_PRESS)
        zRot -= shiftSpeed * 1;
    if (glfwGetKey(window, GLFW_KEY_Q) == GLFW_PRESS)
        zoom += (zoom * (float)ZOOM_SCALE);
    if (glfwGetKey(window, GLFW_KEY_E) == GLFW_PRESS)
        zoom -= (zoom * (float)ZOOM_SCALE);
    
    // timestep adjustment
    // if (glfwGetKey(window, GLFW_KEY_COMMA) == GLFW_PRESS)
    //     timestep -= 0.1f ;
    // if (glfwGetKey(window, GLFW_KEY_PERIOD) == GLFW_PRESS)
    //     timestep += 0.1f;
}
//---------------------------------------


// Triggered when scrollwheel is used
//---------------------------------------
void scroll_callback(GLFWwindow* window, double xoffset, double yoffset)
{   // SCROLL => ZOOM
    zoom += (float)yoffset * (zoom * (float)ZOOM_SCALE);
}
//---------------------------------------


// Triggered when key state changes
//---------------------------------------
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{   // THIS GETS CALLED FOR ALL KEY EVENTS DETECTED
    if (key == GLFW_KEY_F11 && action == GLFW_PRESS)
    {   // CHECKING FOR FULLSCREEN OR NOT
        GLFWmonitor *monitor = glfwGetPrimaryMonitor();
        GLFWmonitor *curMonitor = glfwGetWindowMonitor(window);
        const GLFWvidmode *mode = glfwGetVideoMode(monitor);
        
        if (curMonitor == nullptr)
            glfwSetWindowMonitor(window, monitor, 0, 0, mode->width, mode->height, mode->refreshRate);
        if (curMonitor != nullptr)
            glfwSetWindowMonitor(window, nullptr, 0,0, WIDTH, HEIGHT, 0);
        glfwSwapBuffers(window);
    }
    // BACKSPACE KEY => CLOSE WINDOW
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);
    // Q ENABLES/DISABLES AUTO-ROTATE
    if (key == GLFW_KEY_R && action == GLFW_PRESS) {
        if (!rotateCam) {
            rotateCam = true;
        } else rotateCam = false;
    }
    // CTRL TO SLOW THINGS DOWN
    if (key == GLFW_KEY_LEFT_CONTROL && action == GLFW_PRESS)
        shiftSpeed = 1 * CTRL_FACTOR;
    // COMMA/PERIOD FOR TIMESTEP
    // if (key == GLFW_KEY_COMMA && action == GLFW_PRESS)
    //     m_hDeltaTime -= 0.25f;
    // if (key == GLFW_KEY_PERIOD && action == GLFW_PRESS)
    //     m_hDeltaTime += 10000.25f;
}
//---------------------------------------


// Triggered when the OpenGL window is resized
//---------------------------------------
void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{   // DYNAMICALLY UPDATES VIEWPORT UPON WINDOW RESIZE
    glViewport(0, 0, width, height);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(FOV, (GLfloat)width/(GLfloat)height, 0, V_FAR);
    // TODO: rename to Z_FAR
}
//---------------------------------------



