#include "hip/hip_runtime.h"
//
// Created by quartzar on 23/10/22.
//
#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <random>
#include <fstream>

#include <GL/glew.h> // glut
#include <GLFW/glfw3.h>

// lognormal distribution
#include <map>
#include <iomanip>
// #include <gsl>
#include <gsl/gsl_math.h>
#include <gsl/gsl_cdf.h>

#include "NbKernel_N2.cuh"
#include "CONSTANTS.h"
#include "NbSystemCUDA.cuh"

/////////////////////////////////////////
// █▀█░█▀█░█▄▄░█░▀█▀░█▀▀░█▀█░░░█░█░▀▀█ //
// █▄█░█▀▄░█▄█░█░░█░░██▄░█▀▄░░░▀▄▀░░░█ //
/////////////////////////////////////////

extern __constant__ float softeningSqr;
extern __constant__ float big_G;

//------------PARAMETERS---------------//
NbodyRenderer::RenderMode renderMode = NbodyRenderer::POINTS;
NBodyICConfig sysConfig = NORB_SMALLN_CLUSTER;
NbodyIntegrator integrator = LEAPFROG_VERLET;
NbodyRenderer *renderer = nullptr;
// booleans =>
bool displayEnabled = false;
bool glxyCollision = true;
bool colourMode = true;
bool trailMode = false;
bool outputEnabled = true;
bool outputRealUnits = false;
bool rotateCam = false;
//---------------------------------------q

/////////////////////////////////////////

//---------------------------------------
int main(int argc, char** argv)
{
    //-------------------------
    // CPU data =>
    float4 *m_hPos, *m_hVel, *m_hForce;
    //-------------------------
    // memory transfers =>
    uint m_currentRead, m_currentWrite;
    //-------------------------
    // GPU data =>
    float4 *m_dPos[2], *m_dVel[2], *m_dForce[2];
    //-------------------------
    // OpenGL =>
    GLFWwindow *window = nullptr;
    //-------------------------
    // Timers & benchmarking =>
    auto start = std::chrono::system_clock::now();
    // std::chrono::system_clock::time_point end;
    //-------------------------
    // File output =>
    std::string outputFName = "outputCSV.csv";
    //-------------------------
    // Simulation =>
    int iteration;
    int N_orbitals;
    uint m_p;
    uint m_q;
    N_orbitals = N_BODIES;
    iteration = 0;
    timestep = TIME_STEP;
    m_currentRead = 0;
    m_currentWrite = 1;
    m_p = P;
    m_q = Q;
    zoom = 1;
    //---------------------------------------
    // INITIALISE ARRAYS & ALLOCATE DEVICE STORAGE
    //---------------------------------------
    
    // OLD / HOST
    m_hPos = new float4[N_orbitals]; // x, y, z, mass
    m_hVel = new float4[N_orbitals]; // vx,vy,vz, empty
    m_hForce = new float4[N_orbitals]; // fx, fy, fz, empty
    // NEW / DEVICE
    m_dPos[0] = m_dPos[1] = nullptr;
    m_dVel[0] = m_dVel[1] = nullptr;
    m_dForce[0] = m_dForce[1] = nullptr;
    // set memory for host arrays
    memset(m_hPos, 0, N_orbitals*sizeof(float4));
    memset(m_hVel, 0, N_orbitals*sizeof(float4));
    memset(m_hForce, 0, N_orbitals*sizeof(float4));
    getCUDAError();
    // set memory for device arrays
    allocateNOrbitalArrays(m_dPos,m_dVel, m_dForce, N_orbitals);
    getCUDAError();
    // set device constants
    setDeviceSoftening(SOFTENING);
    setDeviceBigG(1.0f * BIG_G);
    getCUDAError();
    
    //---------------------------------------
    /////////////////////////////////////////
    //---------------------------------------
    
    
    // BEGIN TIMER
    runTimer(start, N_orbitals, true);
    
    // INITIALISE OPENGL
    if (displayEnabled)
    {
        // glutInit(&argc, argv);
        // glutInitDisplayMode(GLUT_RGB | GLUT_DEPTH | GLUT_DOUBLE);
        window = initGL(window);
    }
    
    // PRINT TO FILE
    if (outputEnabled)
    {
        std::ofstream outputFile(outputFName);
        for (int orbital = 0; orbital < N_orbitals; orbital++)
        {
            outputFile << orbital << ","
                       << orbital << "," << orbital << "," << orbital << ","
                       << orbital << "," << orbital << "," << orbital << ","
                       << orbital << "," << orbital << "," << orbital;
            if (orbital != N_orbitals - 1) outputFile << ",";
        }
        outputFile << std::endl;
        for (int orbital = 0; orbital < N_orbitals; orbital++)
        {
            outputFile << "M" << ","
                       <<"x"  << "," << "y"  << "," << "z"  << ","
                       << "vx" << "," << "vy" << "," << "vz" << ","
                       << "fx" << "," << "fy" << "," << "fz";
            if (orbital != N_orbitals - 1) outputFile << ",";
        }
        outputFile << std::endl;
        outputFile.close();
    }
    
    // Randomise Orbitals
    randomiseOrbitals(sysConfig, m_hPos, m_hVel, N_orbitals);
    // Set Initial Forces [only run for solar system, HUGE performance hit]
    if (sysConfig == NORB_CONFIG_SOLAR)
        initialiseForces(m_hPos, m_hForce, N_orbitals);
    
    //---------------------------------------
    // MAIN UPDATE LOOP
    while (iteration <= ITERATIONS)
    {
        if (iteration % 100 == 0)
            std::cout << "\nSTEP =>> " << iteration << std::flush;
    
        if (outputEnabled)
            printToFile(outputFName, iteration, timestep, N_orbitals, m_hPos, m_hVel, m_hForce);
        
        simulate(m_hPos, m_dPos,
                 m_hVel, m_dVel,
                 m_hForce, m_dForce,
                 m_currentRead, m_currentWrite,
                 timestep, N_orbitals, m_p, m_q);
        
        if (displayEnabled && iteration%RENDER_INTERVAL == 0)
        {
            // CHECK FOR INPUT FIRST
            processInput(window);
    
            // CLOSE WINDOW IF ESC PRESSED
            if (glfwWindowShouldClose(window))
            {
                std::cout << "\nPROGRAM TERMINATED BY USER\nEXITING AT STEP " << iteration;
                runTimer(start,  N_orbitals,false);
                finalise(m_hPos, m_dPos,
                         m_hVel, m_dVel,
                         m_hForce, m_dForce);
                glfwTerminate();
                exit(EXIT_SUCCESS);
            }
            
            // render
            renderer->setPositions(reinterpret_cast<float *>(m_hPos));
            renderer->setVelocities(reinterpret_cast<float *>(m_hVel));
            renderer->display(renderMode, zoom, xRot, yRot, zRot, xTrans, yTrans, zTrans, trailMode, colourMode);
    
            glfwSwapBuffers(window);
            // glutSwapBuffers();
            glfwPollEvents();
    
            // set window title to current timestep
            std::string s = std::to_string(iteration);
            const char* cstr = s.c_str();
            glfwSetWindowTitle(window, cstr);
        }
        
        iteration++;
    }
    //---------------------------------------
    
    
    // END TIMER
    runTimer(start,  N_orbitals,false);
    
    // DELETE ARRAYS
    finalise(m_hPos, m_dPos,
             m_hVel, m_dVel,
             m_hForce, m_dForce);
    
    // TERMINATE SUCCESSFULLY
    glfwTerminate();
    exit(EXIT_SUCCESS);
}
//---------------------------------------


// Print to file
//---------------------------------------
void printToFile(const std::string& outputFName, int step, float deltaTime, int N, float4* pos, float4* vel, float4* force)
{
    std::ofstream outputFile;
    outputFile.open(outputFName, std::ios::app); // open file
    
    float mass, xPos, yPos, zPos, xVel, yVel, zVel, xFrc, yFrc, zFrc;
    for (int orbital = 0; orbital < N; orbital++)
    {
        if (outputRealUnits)
        {
            mass = pos[orbital].w * SOLAR_MASS;
            xPos = pos[orbital].x * auTOkm;
            yPos = pos[orbital].y * auTOkm;
            zPos = pos[orbital].z * auTOkm;
            xVel = vel[orbital].x * KMS_TO_AUD;
            yVel = vel[orbital].y * KMS_TO_AUD;
            zVel = vel[orbital].z * KMS_TO_AUD;
            xFrc = force[orbital].x * deltaTime * KMS_TO_AUD;
            yFrc = force[orbital].y * deltaTime * KMS_TO_AUD;
            zFrc = force[orbital].z * deltaTime * KMS_TO_AUD;
        }
        else
        {
            mass = pos[orbital].w;
            xPos = pos[orbital].x;
            yPos = pos[orbital].y;
            zPos = pos[orbital].z;
            xVel = vel[orbital].x;// * (float)KMS_TO_AUD;
            yVel = vel[orbital].y;// * (float)KMS_TO_AUD;
            zVel = vel[orbital].z; // * (float)KMS_TO_AUD;
            xFrc = force[orbital].x * deltaTime;
            yFrc = force[orbital].y * deltaTime;
            zFrc = force[orbital].z * deltaTime;
        }
        
        outputFile << mass << ","
                   << xPos << "," << yPos << "," << zPos << ","
                   << xVel << "," << yVel << "," << zVel << ","
                   << xFrc << "," << yFrc << "," << zFrc;
        if (orbital != N - 1) outputFile << ",";
    }
    outputFile << std::endl; //"\n" doesn't seem to improve performance
    // outputFile.close();

    // outputFile << step << "," << xPos << "," << yPos << "," << zPos << ","
    //            << xVel << "," << yVel << "," << zVel << std::endl;
}
//---------------------------------------


// float lognormalMF(float probability, float zeta, float sigma)
// {
//
//
// }


// IC generator
//---------------------------------------
void randomiseOrbitals(NBodyICConfig config, float4* pos, float4* vel, int N)
{
    using std::uniform_real_distribution;
    std::default_random_engine gen(SEED); // NOLINT(cert-msc51-cpp)
    float totalMass = 0.0;
    
    switch(config) {
        case NORB_SMALLN_CLUSTER: // attempting to implement a lognormal fake-IMF function
        {
            // uniform dist for random number between 0-1
            // plug that into the cumulative lognormal
            // using GNU for now, need to write my own function; ln is fine for now
    
            std::random_device rd;
            std::mt19937 genr(rd());
    
            //  max radius of each cluster
            float radius = 2062; //10e4; // AU
            float offset = -1.f;
            
            // Random number between 0-1
            uniform_real_distribution<double> p(0.0, 1.0);
            uniform_real_distribution<float> xyz(-radius/2.f, radius/2.f);
            uniform_real_distribution<float> v(-2.f/KMS_TO_AUD, 2.f/KMS_TO_AUD);
            
            // Inverse probability lognormal
            const double zeta = 0.1; // solar masses [m_0]
            const double sigma = 0.627; // Chabrier, 2002
            
            std::map<double, double> hist; // for histogram
            for (int i = 0; i < N; i++)
            {
                // how many clusters? how many stars/cluster?
                if ((i /*+ 1*/) % STARS_PER_CLUSTER == 0)
                { // generate new cluster
                    offset = 1.f; // no idea yet
                }
                
                // mass function
                auto prob = p(genr);
                auto mass = gsl_cdf_lognormal_Pinv(prob, zeta, sigma);
                
                // randomised positions based on radius
                float px = xyz(genr);
                float py = xyz(genr);
                float pz = xyz(genr);
                
                // assign positions
                pos[i].x = px + offset * radius;
                pos[i].y = py + offset * radius;
                pos[i].z = pz + offset * radius;
                pos[i].w = float(mass);
                
                // assign velocities [dumb for now]
                vel[i].x = v(genr);
                vel[i].y = v(genr);
                vel[i].z = v(genr);
                // vel[i].x = 0.f;
                // vel[i].y = 0.f;
                // vel[i].z = 0.f;
                vel[i].w = pos[i].w;
                
                
                std::cout << '\n' << mass;
                totalMass += float(mass);
                ++hist[std::round(mass)];
            }
            std::cout << "\nTotal mass: " << totalMass << '\n';
    
            // Inverse probability lognormal
            // double zeta = log10(m_0) - (pow(sigma, 2) / 2);
            // double mass = exp(zeta + (sigma * sqrt(2) * erfinv(2 * probability - 1)));
    
            // Random number between 0-1
            // double px = p(genr);
            // double A = 0.1 / sqrt(2 * PI * pow(sigma, 2));
            // double A = 0.158;// 0.141;
            //
            // double x = A * exp(-1. * (pow(log10(px) - log10(zeta), 2) / (2 * pow(sigma, 2))));
            // // Inverted CDF, also called "quantile function", and specifically for normal dist, "probit function"
            // double mass = zeta + (sigma * sqrt(2) * erfinv(2 * x - 1));
            // using the GNU scientific library
            // std::map<double, double> hist;
            // for(int n=0; n<10000; ++n) {
            //     // ++hist[std::round(p(genr))];
            //     auto prob = p(genr);
            //     auto mass = gsl_cdf_lognormal_Pinv(prob, m_0, sigma);
            //     ++hist[std::round(mass)];
            // }
            for(auto pair : hist) {
                std::cout << '\n' << std::fixed << std::setprecision(1) << std::setw(2)
                          << pair.first << ' ' << std::string(pair.second, '*');
            }
            // for(auto pair : hist) {
            //     std::cout << '\n' << pair.first << ' ' << log(pair.second);
            // }
        }
            break;
        case NORB_CONFIG_BASIC:
        {
            uniform_real_distribution<float> randXPos(-SYS_WIDTH / 2.0, SYS_WIDTH / 2.0);
            uniform_real_distribution<float> randYPos(-SYS_HEIGHT / 2.0, SYS_HEIGHT / 2.0);
            uniform_real_distribution<float> randVel(-INIT_VEL, INIT_VEL);
            uniform_real_distribution<float> randHeight(-SYSTEM_THICKNESS, SYSTEM_THICKNESS);
            uniform_real_distribution<float> randMass(INIT_M_LOWER, INIT_M_HIGHER);
            // returns -1 to 3, so multiply by max mass/3 and clamp between min and max mass
            std::normal_distribution<float> normalDistMass(1, 0.5);
            
            
            // ASSIGNMENT LOOP
            for (int i = 0; i < N_BODIES; i++)
            {
                // getting and clamping normal distribution of mass
                const float mass = normalDistMass(gen) * ((float)INIT_M_HIGHER / 3.f);
                float massClamped;
                if (mass > 1.f * (float)INIT_M_HIGHER)
                {
                    std::cout << "\nbig boi";
                    massClamped = 100000.f;
                }
                else
                    massClamped = std::clamp(mass, (float)INIT_M_LOWER, (float)INIT_M_HIGHER);
                
                // random position assignment
                pos[i].x = randXPos(gen);
                pos[i].y = randYPos(gen);
                pos[i].z = randHeight(gen);
                pos[i].w = massClamped;
    
                // random velocity assignment
                float r = sqrtf(pos[i].x * pos[i].x + pos[i].y * pos[i].y + pos[i].z * pos[i].z);
                vel[i].x = randVel(gen) * (r / pos[i].x);//0.001f;
                vel[i].y = randVel(gen) * (r / pos[i].y);//0.001f;
                vel[i].z = 0.0f;
                vel[i].w = pos[i].w;
        
                totalMass += pos[i].w;
            }
        }
            break;
        case NORB_CONFIG_BASIC_DISK:
        {
            std::cout << "basic disk model to be implemented";
            // also to be implemented
        }
            break;
        case NORB_CONFIG_SHELL:
        {
            uniform_real_distribution<float> randF(0.0f, (float) RAND_MAX);
            uniform_real_distribution<float> randMass(INIT_M_LOWER, INIT_M_HIGHER);
    
            float scale = SYSTEM_SIZE;
            float vScale = scale * (float) VEL_SCALE / (float) KMS_TO_AUD;
            float inner = 2.5f * scale;
            float outer = 4.0f * scale;
    
            pos[0].x = 0.0;
            pos[0].y = 0.0;
            pos[0].z = 0.0;
            pos[0].w = CENTRE_STAR_M;
    
            vel[0].x = 0.0;
            vel[0].y = 0.0;
            vel[0].z = 0.0;
            vel[0].w = CENTRE_STAR_M;
    
    
            int i = 1;
            while (i < N_BODIES) {
                float x, y, z;
                x = randF(gen) / (float) RAND_MAX * 2.0f - 1.0f;
                y = randF(gen) / (float) RAND_MAX * 2.0f - 1.0f;
                z = randF(gen) / (float) RAND_MAX * 2.0f - 1.0f;
        
                float3 point = {x, y, z};
                float len = normalise(point);
                if (len > 1)
                    continue;
    
                pos[i].x = point.x * (inner + (outer - inner) * randF(gen) / (float) RAND_MAX);
                pos[i].y = point.x * (inner + (outer - inner) * randF(gen) / (float) RAND_MAX);
                pos[i].z = point.x * (inner + (outer - inner) * randF(gen) / (float) RAND_MAX);
                pos[i].w = randMass(gen);
                
        
                x = 0.0f;
                y = 0.0f;
                z = 1.0f;
        
                float3 axis = {x, y, z};
                normalise(axis);
        
                if (1 - dot(point, axis) < 1e-6) {
                    axis.x = point.y;
                    axis.y = point.x;
                    normalise(axis);
                }
                float3 vv = {pos[i].x, pos[i].y, pos[i].z};
                vv = cross(vv, axis);
                vel[i].x = vv.x * vScale;
                vel[i].y = vv.y * vScale;
                vel[i].z = vv.z * vScale;
                vel[i].w = pos[i].w;
        
                i++;
            }
        }
            break;
        case NORB_CONFIG_EXPAND:
        {
            uniform_real_distribution<float> randF(0.0f, (float) RAND_MAX);
            uniform_real_distribution<float> randMass(INIT_M_LOWER, INIT_M_HIGHER);
    
            float scale = SYSTEM_SIZE * std::max(1.0f, (float)N / (1024.f));
            float vScale = scale * (float) VEL_SCALE / (float) KMS_TO_AUD;
            
            for (int i = 0; i < N;)
            {
                float3 point;
                point.x = randF(gen) / (float) RAND_MAX * 2.0f - 1.0f;
                point.y = randF(gen) / (float) RAND_MAX * 2.0f - 1.0f;
                point.z = randF(gen) / (float) RAND_MAX * 2.0f - 1.0f;
                
                float lengthSq = dot(point, point);
                if (lengthSq > 1)
                    continue;
    
                pos[i].x = point.x * scale;
                pos[i].y = point.y * scale;
                pos[i].z = point.z * scale;
                pos[i].w = randMass(gen);
                vel[i].x = point.x * vScale; //* float(PI)/180 * lengthSq;
                vel[i].y = point.y * vScale;
                vel[i].z = point.z * vScale;
                vel[i].w = pos[i].w;
                
                i++;
            }
            
        }
            break;
        case NORB_CONFIG_ADV_DISK:
        {
            // uniform_real_distribution<float> randF(0.0f, (float) RAND_MAX);
            // uniform_real_distribution<float> randMass(0.0, 5);
            uniform_real_distribution<float> randMassInner(ADVD_M_INNER_MIN, ADVD_M_INNER_MAX);
            // uniform_real_distribution<float> randMassOuter(INIT_M_LOWER, INIT_M_HIGHER);
            
            pos[0].x = 0.0;
            pos[0].y = 0.0;
            pos[0].z = 0.0;
            pos[0].w = ADVD_CENTRE_M;
    
            vel[0].x = 0.0;
            vel[0].y = 0.0;
            vel[0].z = 0.0;
            vel[0].w = ADVD_CENTRE_M;
    
            float c      = ADVD_C_INNER; // flatness
            float mass   = randMassInner(gen);
            // float mass = randMassInner(gen);
            float radius = ADVD_R_INNER;
            
            int start;
            if (glxyCollision)
            {
                pos[1].x = 1000.0;
                pos[1].y = 500.0;
                pos[1].z = -10000.0;
                pos[1].w = ADVD_G2_MASS;
    
                vel[1].x = -0.1;
                vel[1].y = 0.0;
                vel[1].z = 1.0;
                vel[1].w = ADVD_G2_MASS;
                start = 2;
            }
            else
                start = 1;
            for (int i = start;i < N; i++)
            {
                if (i == N - ADVD_OUTER_N) {
                    c = ADVD_C_OUTER;
                    mass = ADVD_M_OUTER;
                    // mass = randMassOuter(gen) * 100.0f;
                    radius = ADVD_R_OUTER;
                }
                
                float3 position;
                while (true)
                {
                    position.x = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
                    position.y = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
                    
                    if (position.y >= -1.0f * sqrtf(1.0f - powf(position.x, 2.0f))
                        && position.y <= sqrtf(1.0f - powf(position.x, 2.0f)))
                        break;
                }
                
                float zPosMax = sqrtf(c * (1.0f - powf(position.x, 2.0f)
                        -powf(position.y, 2.0f)));
                float zPosMin = -1.0f * zPosMax;
                float zPosRand = rand() / (float) RAND_MAX;
                position.z = (zPosMax - zPosMin) * zPosRand + zPosMin;
    
                position.x *= radius;
                position.y *= radius;
                position.z *= radius;
                
                float m = position.y / position.x;
                m = -1.0f / m;
                float b = position.y - position.x * m;
                
                float3 velocity;
                // float vel_m = sqrtf(((float)BIG_G * (1e6f + mass * 1.2e-6f)) /
                //                     sqrtf(position.x*position.x + position.y*position.y + position.z*position.z));
                float vel_m = sqrtf(((float)BIG_G * (ADVD_CENTRE_M + mass * 1.2e2f)) /
                                    sqrtf(position.x * position.x + position.y * position.y + position.z * position.z));
                
                if (position.y > 0)
                {
                    velocity = {-1.0f * (radius / 2.0f), (position.x - radius / 2.0f) * m + b - position.y, 0};
                    vel_m /= sqrtf(velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z);
                    velocity.x *= vel_m;
                    velocity.y *= vel_m;
                    velocity.z *= vel_m;
                    
                }
                else
                {
                    velocity = {(radius / 2.0f), (position.x + radius / 2.0f) * m + b - position.y, 0};
                    vel_m /= sqrtf(velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z);
                    velocity.x *= vel_m;
                    velocity.y *= vel_m;
                    velocity.z *= vel_m;
                }
                
                float pScale = 1.0f;
                pos[i].x = position.x * pScale;
                pos[i].y = position.y * pScale;
                pos[i].z = position.z * pScale;
                pos[i].w = mass;
    
                vel[i].x = velocity.x;
                vel[i].y = velocity.y;
                vel[i].z = velocity.z;
                vel[i].w = mass;
                
                // std::cout << "\n " << velocity[i].x << " " << velocity[i].y << " " << velocity[i].z;
    
            }
        }
            break;
        case NORB_CONFIG_ADV_DISK_COLLSION:
        {
            // hi
            uniform_real_distribution<float> randMassInner(ADVD_M_INNER_MIN, ADVD_M_INNER_MAX);
    
            pos[0].x = 0.0;
            pos[0].y = 0.0;
            pos[0].z = 0.0;
            pos[0].w = ADVD_CENTRE_M;
    
            vel[0].x = 0.0;
            vel[0].y = 0.0;
            vel[0].z = 0.0;
            vel[0].w = ADVD_CENTRE_M;
    
            pos[N / 2].x = ADVD_G2_X;
            pos[N / 2].y = ADVD_G2_Y;
            pos[N / 2].z = ADVD_G2_Z;
            pos[N / 2].w = ADVD_CENTRE_M;
    
            vel[N / 2].x = -1.f * ADVD_G2_VX;
            vel[N / 2].y = -1.f * ADVD_G2_VY;
            vel[N / 2].z = -1.f * ADVD_G2_VZ;
            vel[N / 2].w = ADVD_CENTRE_M;
    
            float c      = ADVD_C_INNER; // flatness
            float mass;//   = randMassInner(gen);
            // float mass = randMassInner(gen);
            float radius = ADVD_R_INNER;
    
            int start = 1;
            for (int i = start;i < N/2; i++)
            {
                mass = randMassInner(gen);
                if (i == N/2 - ADVD_OUTER_N) {
                    c = ADVD_C_OUTER;
                    mass = ADVD_M_OUTER;
                    // mass = randMassOuter(gen) * 100.0f;
                    radius = ADVD_R_OUTER;
                }
        
                float3 position;
                while (true)
                {
                    position.x = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
                    position.y = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
            
                    if (position.y >= -1.0f * sqrtf(1.0f - powf(position.x, 2.0f))
                        && position.y <= sqrtf(1.0f - powf(position.x, 2.0f)))
                        break;
                }
        
                float zPosMax = sqrtf(c * (1.0f - powf(position.x, 2.0f)
                                           -powf(position.y, 2.0f)));
                float zPosMin = -1.0f * zPosMax;
                float zPosRand = rand() / (float) RAND_MAX;
                position.z = (zPosMax - zPosMin) * zPosRand + zPosMin;
    
                position.x *= radius;
                position.y *= radius;
                position.z *= radius;
        
                float m = position.y / position.x;
                m = -1.0f / m;
                float b = position.y - position.x * m;
        
                float3 velocity;
                // float vel_m = sqrtf(((float)BIG_G * (1e6f + mass * 1.2e-6f)) /
                //                     sqrtf(position.x*position.x + position.y*position.y + position.z*position.z));
                float vel_m = sqrtf(((float)BIG_G * (ADVD_CENTRE_M + mass * 1.2e2f)) /
                                    sqrtf(position.x * position.x + position.y * position.y + position.z * position.z));
        
                if (position.y > 0)
                {
                    velocity = {-1.0f * (radius / 2.0f), (position.x - radius / 2.0f) * m + b - position.y, 0};
                    vel_m /= sqrtf(velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z);
                    velocity.x *= vel_m;
                    velocity.y *= vel_m;
                    velocity.z *= vel_m;
            
                }
                else
                {
                    velocity = {(radius / 2.0f), (position.x + radius / 2.0f) * m + b - position.y, 0};
                    vel_m /= sqrtf(velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z);
                    velocity.x *= vel_m;
                    velocity.y *= vel_m;
                    velocity.z *= vel_m;
                }
        
                float pScale = 1.0f;
                pos[i].x = position.x * pScale;
                pos[i].y = position.y * pScale;
                pos[i].z = position.z * pScale;
                pos[i].w = mass;
    
                vel[i].x = velocity.x;
                vel[i].y = velocity.y;
                vel[i].z = velocity.z;
                vel[i].w = mass;
                }
    
            c      = ADVD_C_INNER; // flatness
            mass   = randMassInner(gen);
            // float mass = randMassInner(gen);
            radius = ADVD_R_INNER;
            start = N/2 + 1;
            for (int i = start;i < N; i++)
            {
                mass = randMassInner(gen);
                if (i == N - ADVD_OUTER_N) {
                    c = ADVD_C_OUTER;
                    mass = ADVD_M_OUTER;
                    // mass = randMassOuter(gen) * 100.0f;
                    radius = ADVD_R_OUTER;
                }
        
                float3 position;
                while (true)
                {
                    position.x = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
                    position.y = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
            
                    if (position.y >= -1.0f * sqrtf(1.0f - powf(position.x, 2.0f))
                        && position.y <= sqrtf(1.0f - powf(position.x, 2.0f)))
                        break;
                }
        
                float zPosMax = sqrtf(c * (1.0f - powf(position.x, 2.0f)
                                           -powf(position.y, 2.0f)));
                float zPosMin = -1.0f * zPosMax;
                float zPosRand = rand() / (float) RAND_MAX;
                position.z = (zPosMax - zPosMin) * zPosRand + zPosMin;
    
                position.x *= radius;
                position.y *= radius;
                position.z *= radius;
        
                float m = position.y / position.x;
                m = -1.0f / m;
                float b = position.y - position.x * m;
        
                float3 velocity;
                // float vel_m = sqrtf(((float)BIG_G * (1e6f + mass * 1.2e-6f)) /
                //                     sqrtf(position.x*position.x + position.y*position.y + position.z*position.z));
                float vel_m = sqrtf(((float)BIG_G * (ADVD_CENTRE_M + mass * 1.2e2f)) /
                                    sqrtf(position.x * position.x + position.y * position.y + position.z * position.z));
        
                if (position.y > 0)
                {
                    velocity = {-1.0f * (radius / 2.0f), (position.x - radius / 2.0f) * m + b - position.y, 0};
                    vel_m /= sqrtf(velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z);
                    velocity.x *= vel_m;
                    velocity.y *= vel_m;
                    velocity.z *= vel_m;
            
                }
                else
                {
                    velocity = {(radius / 2.0f), (position.x + radius / 2.0f) * m + b - position.y, 0};
                    vel_m /= sqrtf(velocity.x * velocity.x + velocity.y * velocity.y + velocity.z * velocity.z);
                    velocity.x *= vel_m;
                    velocity.y *= vel_m;
                    velocity.z *= vel_m;
                }
    
    
                pos[i].x = position.z + (float)ADVD_G2_X;
                pos[i].y = position.y + (float)ADVD_G2_Y;
                pos[i].z = position.x + (float)ADVD_G2_Z;
                pos[i].w = mass;
    
                vel[i].x = velocity.z - (float)ADVD_G2_VX;
                vel[i].y = velocity.y - (float)ADVD_G2_VY;
                vel[i].z = velocity.x - (float)ADVD_G2_VZ;
                vel[i].w = mass;
            }
            
        }
            break;
        case NORB_CONFIG_SOLAR:
        {
            int i = 0;
            // The Sun
            pos[i].x = pos[i].y = pos[i].z = 0.f;
            pos[i].w = 1.f;
    
            vel[i].x = vel[i].y = vel[i].z = 0.f;
            vel[i].w = 1.f;
    
            // Earth
            pos[++i].x = 1.f;
            pos[i].y = 0.f;
            pos[i].z = 0.f;
            pos[i].w = 3.00273e-6f;// 2.9861e-6f;
    
            vel[i].x = 0.f;
            vel[i].y = 29.795f / KMS_TO_AUD;//29.78f / (float)KMS_TO_AUD;
            vel[i].z = 0.f;
            vel[i].w = 3.00273e-6f;
    
            // Mercury
            pos[++i] = {.387f, 0.f, 0.f, 1.651e-7f};
            vel[i]   = {0.f, 47.36f/KMS_TO_AUD, 0.f, 1.651e-7f};

            // Venus
            pos[++i].x = 0.723f;
            pos[i].y = 0.f;
            pos[i].z = 0.f;
            pos[i].w = 2.447e-6f;

            vel[i].x = 0.f;
            vel[i].y = 35.02f / KMS_TO_AUD;
            vel[i].z = 0.f;
            vel[i].w = 2.447e-6f;

            // Mars
            pos[++i] = {1.524f, 0.f, 0.f, 3.213e-7f};
            vel[i]   = {0.f, 24.07f/KMS_TO_AUD, 0.f, 3.213e-7f};
            
        }
            break;
    }
    std::cout << "\nTOTAL MASS ->> " << totalMass;
}
//---------------------------------------


// Print to file
//---------------------------------------
void initialiseForces(float4* pos, float4* force, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            if (i == j)
                continue;
            
            float3 r;
    
            // r_ij -> AU [distance]
            r.x = pos[j].x - pos[i].x;
            r.y = pos[j].y - pos[i].y;
            r.z = pos[j].z - pos[i].z;
    
            // distance squared == dot(r_ij, r_ij) + softening^2
            float distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
            distSqr += SOFTENING * SOFTENING;
    
            // inverse distance cubed == 1 / distSqr^(3/2) [fastest method]
            float distSixth = distSqr * distSqr * distSqr;
            float invDistCube = 1.0f / sqrtf(distSixth);
    
            // force = mass_j * inverse distance cube
            float f = pos[j].w * invDistCube;
    
            // acceleration = acceleration_i + force * r_ij
            force[i].x += r.x * f * (float)BIG_G;
            force[i].y += r.y * f * (float)BIG_G;
            force[i].z += r.z * f * (float)BIG_G;
        }
    }
}
//---------------------------------------



//---------------------------------------
// MAIN UPDATE LOOP
//---------------------------------------
void simulate(float4* m_hPos, float4* m_dPos[2],
              float4* m_hVel, float4* m_dVel[2],
              float4* m_hForce, float4* m_dForce[2],
              uint m_currentRead, uint m_currentWrite,
              float deltaTime, int N, uint m_p, uint m_q)
{
    // set pos,vel -> update -> get pos,vel ~@ repeat
    
    // Send data to device first
    copyDataToDevice(m_dPos[m_currentRead], m_hPos, N);
    copyDataToDevice(m_dVel[m_currentRead], m_hVel, N);
    copyDataToDevice(m_dForce[m_currentRead], m_hForce, N);


    // Do the thing
    deployToGPU(m_dPos[m_currentRead], m_dPos[m_currentWrite],
                m_dVel[m_currentRead], m_dVel[m_currentWrite],
                m_dForce[m_currentRead], m_dForce[m_currentWrite],
                deltaTime, N, m_p, m_q);
    // Swap read and write
    std::swap(m_currentRead, m_currentWrite);

    // hipDeviceSynchronize();
    
    // Retrieve data from device
    copyDataToHost(m_hPos, m_dPos[m_currentRead], N);
    copyDataToHost(m_hVel, m_dVel[m_currentRead], N);
    copyDataToHost(m_hForce, m_dForce[m_currentRead], N);

    // Retrieve any CUDA errors and output
    getCUDAError();
}
//---------------------------------------


// CUDA error check
//---------------------------------------
void getCUDAError()
{
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess) std::cout << "\nCUDA error:%s\n" << hipGetErrorString(hipError_t);
}
//---------------------------------------


// Finalise & delete arrays TODO: reimplement this
//---------------------------------------
void finalise(float4* m_hPos, float4* m_dPos[2],
              float4* m_hVel, float4* m_dVel[2],
              float4* m_hForce, float4* m_dForce[2])
{
    delete [] m_hPos;
    delete [] m_hVel;
    delete [] m_hForce;
    
    deleteNOrbitalArrays(m_dPos, m_dVel, m_dForce);
}
//---------------------------------------


// A nice little normalisation function
//---------------------------------------
float normalise(float3& vector)
{
    float dist = sqrtf(vector.x*vector.x + vector.y*vector.y + vector.z*vector.z);
    if (dist > 1e-6)
    {
        vector.x /= dist;
        vector.y /= dist;
        vector.z /= dist;
    }
    return dist;
}
//---------------------------------------



//////////////////////////////////////////////////////
// █▀▀ █▀█ █░█   █▀▀ █░█ █▄░█ █▀▀ ▀█▀ █ █▀█ █▄░█ █▀ //
// █▄█ █▀▀ █▄█   █▀░ █▄█ █░▀█ █▄▄ ░█░ █ █▄█ █░▀█ ▄█ //
//////////////////////////////////////////////////////

extern "C"
{

// Send softening_sqr value to device
//---------------------------------------
void setDeviceSoftening(float softening)
{
    float softeningSq = softening * softening;
    
    hipMemcpyToSymbol(HIP_SYMBOL(softeningSqr), &softeningSq, sizeof(float),0);
}
//---------------------------------------


// Send gravitational constant to device
//---------------------------------------
void setDeviceBigG(float G)
{
    hipMemcpyToSymbol(HIP_SYMBOL(big_G), &G, sizeof(float),0);
}
//---------------------------------------


// Allocate device memory for variables
//---------------------------------------
void allocateNOrbitalArrays(float4* pos[2], float4* vel[2], float4* force[2],  int N)
{
    // memory size for device allocation
    uint memSize = sizeof(float4) * N;
    // uint fMemSize = sizeof(float3) * N;
    
    hipMalloc((void**)&pos[0], memSize);
    hipMalloc((void**)&pos[1], memSize);
    hipMalloc((void**)&vel[0], memSize);
    hipMalloc((void**)&vel[1], memSize);
    hipMalloc((void**)&force[0], memSize);
    hipMalloc((void**)&force[1], memSize);
}
//---------------------------------------


// De-allocate device memory variables
//---------------------------------------
void deleteNOrbitalArrays(float4* pos[2], float4* vel[2], float4* force[2])
{
    hipFree((void**)pos[0]);
    hipFree((void**)pos[1]);
    hipFree((void**)vel[0]);
    hipFree((void**)vel[1]);
    hipFree((void**)force[0]);
    hipFree((void**)force[1]);
}
//---------------------------------------


// Copy data from host[CPU] ->> device[GPU]
//---------------------------------------
void copyDataToDevice(float4* device, const float4* host, int N)
{
    uint memSize = sizeof(float4) * N;
    hipMemcpy(device, host, memSize, hipMemcpyHostToDevice);
    getCUDAError();
}
//---------------------------------------


// Copy data from device[GPU] ->> host[CPU]
//---------------------------------------
void copyDataToHost(float4* host, const float4* device, int N)
{
    uint memSize = sizeof(float4) * N;
    hipMemcpy(host, device, memSize, hipMemcpyDeviceToHost);
    getCUDAError();
}
//---------------------------------------


// Initiates GPU kernel computations every iteration
//---------------------------------------
void deployToGPU(float4* oldPos, float4* newPos,
                 float4* oldVel, float4* newVel,
                 float4* oldForce, float4* newForce,
                 float deltaTime, int N, uint p, uint q)
{
    uint shMemSize = p * q * sizeof(float4);
    
    // thread and grid time :D
    dim3 threads(p, q, 1);
    dim3 grid(N / p, 1, 1);
    
    // DEPLOY TODO: removed feature
    /*If multithreading is enabled (i.e. q>1 | multiple threads per
     * body) then the more complicated code is executed (using bool template
     * over in the kernel), and if it is not, then the simpler code is executed*/

    switch(integrator)
    {
        case LEAPFROG_VERLET:
        default:
        {
            integrateNOrbitals<<<grid, threads, shMemSize
            >>>(oldPos, newPos, oldVel, newVel, oldForce, newForce, deltaTime, N);
        }
        break;
        case KICK_DRIFT_VERLET:
        {
            initHalfKickForces<<<grid, threads, shMemSize
            >>>(oldPos, newPos, oldVel, newVel, oldForce, newForce, deltaTime, N);
            hipDeviceSynchronize();
            fullKickForces<<<grid, threads, shMemSize
            >>>(oldPos, newPos, oldVel, newVel, oldForce, newForce, deltaTime, N);
        }
        break;
    }
}
//---------------------------------------
}

// MISC FUNCTIONS

// Timer, very simple
//---------------------------------------
void runTimer(std::chrono::system_clock::time_point start,
              int N_orbitals, bool init)
{
    if (init)
    {
        start = std::chrono::system_clock::now();
        std::time_t start_time = std::chrono::system_clock::to_time_t(start);
        std::cout << "Starting Simulation at ->> " << std::ctime(&start_time)
                  << "For N == " << N_orbitals << " || Iterations == " << ITERATIONS;
    }
    else // end
    {
        auto end = std::chrono::system_clock::now();
        std::chrono::duration<double> elapsed_seconds = end-start;
        std::time_t end_time = std::chrono::system_clock::to_time_t(end);
        std::cout << "\nFinished Computation at ->> " << std::ctime(&end_time)
                  << "Elapsed Time : " << elapsed_seconds.count() << "s"
                  << " for N = " << N_orbitals << std::endl;
    }
}
//---------------------------------------


// Initialise OpenGL for particle rendering
//---------------------------------------
GLFWwindow* initGL(GLFWwindow *window)
{
    if(!glewInit())
    {
        std::cout << "\nCritical OpenGL error ::\nFailed to initialise GLEW\nTERMINATING";
        glfwTerminate();
        exit(EXIT_FAILURE);
    }
    if (!glfwInit())
    {   // SAFETY CHECK
        std::cout << "\nCritical OpenGL error ::\nFailed to initialise GLFW\nTERMINATING";
        glfwTerminate();
        exit(EXIT_FAILURE);
    }
    
    // CREATE WINDOW IN WINDOWED MODE
    glfwWindowHint(GLFW_RESIZABLE, GL_TRUE);
    window = glfwCreateWindow(WIDTH, HEIGHT, "orbiterV6", nullptr, nullptr);
    
    if (!window)
    {   // SAFETY CHECK
        std::cout << "\n Critical OpenGL error ::\nFailed to open GLFW window\nTERMINATING";
        glfwTerminate();
        exit (EXIT_FAILURE);
    }
    // CALLBACKS
    glfwSetFramebufferSizeCallback(window, framebuffer_size_callback); // -> viewport
    glfwSetKeyCallback(window, key_callback); // -> key input
    glfwSetScrollCallback(window, scroll_callback); // -> scroll input
    
    // set window context | synchronise to refresh rate with swapinterval
    glfwMakeContextCurrent(window);
    
    // SET THE VIEWPORT
    glViewport(0, 0, WIDTH, HEIGHT);
    // SET THE PROJECTION TRANSFORM
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(FOV, (GLfloat)WIDTH/(GLfloat)HEIGHT, 0, V_FAR); // TODO: rename to Z_FAR
    
    // PREPARE WINDOW
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GL_LEQUAL); // experimental
    glClearColor(0.0, 0.0, 0.0, 1.0);
    
    // PREPARE RENDERER
    renderer = new NbodyRenderer;
    
    // TODO: add GL error check here
    return window;
}
//---------------------------------------


// A nice little vector cross product function
//---------------------------------------
float3 cross(float3 v0, float3 v1)
{
    float3 rt;
    rt.x = v0.y*v1.z-v0.z*v1.y;
    rt.y = v0.z*v1.x-v0.x*v1.z;
    rt.z = v0.x*v1.y-v0.y*v1.x;
    return rt;
}
//---------------------------------------


// A nice little vector dot product function
//---------------------------------------
float dot(float3 v0, float3 v1)
{
    return v0.x*v1.x+v0.y*v1.y+v0.z*v1.z;
}
//---------------------------------------


// Processes user input for sim control
//---------------------------------------
void processInput(GLFWwindow *window)
{
    // if (glfwGetKey(window, GLFW_KEY_ESCAPE) == GLFW_PRESS)
    
    if (glfwGetKey(window, GLFW_KEY_LEFT_SHIFT) == GLFW_PRESS)
        shiftSpeed = 1 * SHIFT_FACTOR;
    if (glfwGetKey(window, GLFW_KEY_LEFT_SHIFT) == GLFW_RELEASE)
        shiftSpeed = 1;
    if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS)
        zTrans += shiftSpeed * MOVE_SPEED * 1.0f;
    if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS)
        zTrans -= shiftSpeed * MOVE_SPEED * 1.0f;
    if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS)
        xTrans += shiftSpeed * MOVE_SPEED;
    if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS)
        xTrans -= shiftSpeed * MOVE_SPEED;
    if (glfwGetKey(window, GLFW_KEY_SPACE) == GLFW_PRESS)
        yTrans -= shiftSpeed * MOVE_SPEED;
    if (glfwGetKey(window, GLFW_KEY_LEFT_CONTROL) == GLFW_PRESS)
        yTrans += shiftSpeed * MOVE_SPEED;
    if (glfwGetKey(window, GLFW_KEY_LEFT) == GLFW_PRESS)
        xRot += shiftSpeed * 1;
    if (glfwGetKey(window, GLFW_KEY_RIGHT) == GLFW_PRESS)
        xRot -= shiftSpeed * 1;
    if (glfwGetKey(window, GLFW_KEY_UP) == GLFW_PRESS)
        yRot += shiftSpeed * 1;
    if (glfwGetKey(window, GLFW_KEY_DOWN) == GLFW_PRESS)
        yRot -= shiftSpeed * 1;
    if (glfwGetKey(window, GLFW_KEY_Z) == GLFW_PRESS)
        zRot += shiftSpeed * 1;
    if (glfwGetKey(window, GLFW_KEY_X) == GLFW_PRESS)
        zRot -= shiftSpeed * 1;
    if (glfwGetKey(window, GLFW_KEY_Q) == GLFW_PRESS)
        zoom += (zoom * (float)ZOOM_SCALE);
    if (glfwGetKey(window, GLFW_KEY_E) == GLFW_PRESS)
        zoom -= (zoom * (float)ZOOM_SCALE);
    
    // timestep adjustment
    if (glfwGetKey(window, GLFW_KEY_COMMA) == GLFW_PRESS)
        timestep -= 0.1f ;
    if (glfwGetKey(window, GLFW_KEY_PERIOD) == GLFW_PRESS)
        timestep += 0.1f;
}
//---------------------------------------


// Triggered when scrollwheel is used
//---------------------------------------
void scroll_callback(GLFWwindow* window, double xoffset, double yoffset)
{   // SCROLL => ZOOM
    zoom += (float)yoffset * (zoom * (float)ZOOM_SCALE);
}
//---------------------------------------


// Triggered when key state changes
//---------------------------------------
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{   // THIS GETS CALLED FOR ALL KEY EVENTS DETECTED
    if (key == GLFW_KEY_F11 && action == GLFW_PRESS)
    {   // CHECKING FOR FULLSCREEN OR NOT
        GLFWmonitor *monitor = glfwGetPrimaryMonitor();
        GLFWmonitor *curMonitor = glfwGetWindowMonitor(window);
        const GLFWvidmode *mode = glfwGetVideoMode(monitor);
        
        if (curMonitor == nullptr)
            glfwSetWindowMonitor(window, monitor, 0, 0, mode->width, mode->height, mode->refreshRate);
        if (curMonitor != nullptr)
            glfwSetWindowMonitor(window, nullptr, 0,0, WIDTH, HEIGHT, 0);
        glfwSwapBuffers(window);
    }
    // BACKSPACE KEY => CLOSE WINDOW
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
        glfwSetWindowShouldClose(window, true);
    // Q ENABLES/DISABLES AUTO-ROTATE
    if (key == GLFW_KEY_R && action == GLFW_PRESS) {
        if (!rotateCam) {
            rotateCam = true;
        } else rotateCam = false;
    }
    // COMMA/PERIOD FOR TIMESTEP
    // if (key == GLFW_KEY_COMMA && action == GLFW_PRESS)
    //     timestep -= 0.25f;
    // if (key == GLFW_KEY_PERIOD && action == GLFW_PRESS)
    //     timestep += 0.25f;
}
//---------------------------------------


// Triggered when the OpenGL window is resized
//---------------------------------------
void framebuffer_size_callback(GLFWwindow* window, int width, int height)
{   // DYNAMICALLY UPDATES VIEWPORT UPON WINDOW RESIZE
    glViewport(0, 0, width, height);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(FOV, (GLfloat)width/(GLfloat)height, 0, V_FAR);
    // TODO: rename to Z_FAR
}
//---------------------------------------



